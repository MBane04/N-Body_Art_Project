#include "hip/hip_runtime.h"
/* 
    this file contains the followinf functions:
        float4 centerOfMass();
        float4 linearVelocity();
        void zeroOutSystem();
        void getForces(Body* bodies, float mass, float G, float H, float Epsilon, float drag, float dt, int n);
        void nBody();
        
*/

#include "./header.h"

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < numBodies; i++)
	{
    	centerOfMass.x += bodies[i].pos.x*MassOfBody;
		centerOfMass.y += bodies[i].pos.y*MassOfBody;
		centerOfMass.z += bodies[i].pos.z*MassOfBody;
		totalMass += MassOfBody;
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float totalMass;
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < numBodies; i++)
	{
    	linearVelocity.x += bodies[i].vel.x*MassOfBody;
		linearVelocity.y += bodies[i].vel.y*MassOfBody;
		linearVelocity.z += bodies[i].vel.z*MassOfBody;
		totalMass += MassOfBody;
	}
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	
	return(linearVelocity);
}

void zeroOutSystem()
{
	float4 pos, vel;
	pos = centerOfMass();
	vel = linearVelocity();
		
	for(int i = 0; i < numBodies; i++)
	{
		bodies[i].pos.x -= pos.x;
		bodies[i].pos.y -= pos.y;
		bodies[i].pos.z -= pos.z;
		
		bodies[i].vel.x -= vel.x;
		bodies[i].vel.y -= vel.y;
		bodies[i].vel.z -= vel.z;
	}
}

void getForces(Body* bodies, float mass, float G, float H, float Epsilon, float drag, float dt, int n)
{
	float dx, dy, dz, d2, d;
    float forceMag;
    float inOut;
	float kSphereReduction = 0.5;
	float dvx, dvy, dvz;
	float kSphere = 10000;

    // Initialize forces to zero
    for (int i = 0; i < n; i++)
    {
        bodies[i].force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }
    
    // Calculate forces
    for (int i = 0; i < n; i++)
    {
        if(bodies[i].movement != 1) //if the body is not still (movement 1 is still)
        {

        
            for (int j = i + 1; j < n; j++)
            {
                dx = bodies[j].pos.x - bodies[i].pos.x;
                dy = bodies[j].pos.y - bodies[i].pos.y;
                dz = bodies[j].pos.z - bodies[i].pos.z;
                d2 = dx * dx + dy * dy + dz * dz + Epsilon;
                d = sqrt(d2);
                if (d < 1e-6) 
                {
                    fprintf(stderr, "Warning: Small distance in force calculation, skipping\n");
                    continue;
                }
                //forceMag = (G * mass * mass) / d2 - (H * mass * mass) / (d2 * d2); // gravitational force
                forceMag = 0.0; //No force between bodies. Each body acts individually.

                float3 force = make_float3(forceMag * dx / d,
                                        forceMag * dy / d,
                                        forceMag * dz / d);

                if(bodies[i].isSolid ^ bodies[j].isSolid) //bitwise XOR. If one is solid and the other is not, and only then, do the following.
                {
                    float combinedDiamter = bodies[i].radius + bodies[j].radius;
                    if(d < combinedDiamter) //if the balls touch. i.e if the distance betweeen < both radii
                    {
                        
                        dvx = bodies[j].vel.x - bodies[i].vel.x;
                        dvy = bodies[j].vel.y - bodies[i].vel.y;
                        dvz = bodies[j].vel.z - bodies[i].vel.z;
                        inOut = dx*dvx + dy*dvy + dz*dvz;
                        if(inOut < 0.0) forceMag = kSphere*(combinedDiamter - d); // If inOut is negative the sphere are converging.
                        else forceMag = kSphereReduction*kSphere*(combinedDiamter - d); // If inOut is positive the sphere are diverging.
                        
                        // Doling out the force in the proper perfortions using unit vectors.
                        bodies[i].force.x -= forceMag*(dx/d);
                        bodies[i].force.y -= forceMag*(dy/d);
                        bodies[i].force.y -= forceMag*(dz/d);
                        // A force on me causes the opposite force on you. 
                        bodies[j].force.x += forceMag*(dx/d);
                        bodies[j].force.y += forceMag*(dy/d);
                        bodies[j].force.z += forceMag*(dz/d);
                    }
                }

                bodies[i].force.x += force.x;
                bodies[i].force.y += force.y;
                bodies[i].force.z += force.z;

                bodies[j].force.x -= force.x;
                bodies[j].force.y -= force.y;
                bodies[j].force.z -= force.z;
            }
        }
    }
	
}

void nBody()
{
    if (Pause != 1)
    {
        // Update positions and velocities
        for (int i = 0; i < numBodies; i++)
        {
            if (bodies[i].movement == 2) // sinusoidal
            {
                float frequency = 1.0f; // Adjust this value to change the period of the sine wave
                float amplitude = 0.2f; // Adjust this value to change the amplitude of the sine wave
    
                bodies[i].pos.x += bodies[i].vel.x * Dt;
                bodies[i].pos.y = bodies[i].initialY + amplitude * sin(frequency * bodies[i].pos.x);
            }
            else if (bodies[i].movement == 3) // circular
            {
                float angularVelocity = 2.0f; // Adjust this value to change the angular velocity of the circle

                float angle = bodies[i].circle.z + angularVelocity * RunTime;
                bodies[i].pos.x = bodies[i].circle.x + bodies[i].circle.w * cos(angle);
                bodies[i].pos.y = bodies[i].circle.y + bodies[i].circle.w * sin(angle);
            }
            else if (bodies[i].movement == 4) // Oscillation movement
            {
                float time = RunTime; // Use the elapsed time for smooth oscillation
                float frequency = 1.0f; // Adjust this value to change the frequency of the oscillation
                float amplitude = bodies[i].oscillationAmplitude; // Use the amplitude set for the body
                float angle = bodies[i].oscillationAngle; // Use the angle set for the body

                // Calculate the new position using a sine function
                bodies[i].pos.x = bodies[i].initialX + amplitude * cos(angle) * sin(frequency * time);
                bodies[i].pos.y = bodies[i].initialY + amplitude * sin(angle) * sin(frequency * time);
            }
            else
            {
                // Update position based on velocity for other movement types
                bodies[i].pos.x += bodies[i].vel.x * Dt;
                bodies[i].pos.y += bodies[i].vel.y * Dt;
                bodies[i].pos.z += bodies[i].vel.z * Dt;
            }
        }

        DrawTimer++;
        if (DrawTimer == DrawRate)
        {
            drawPicture();
            DrawTimer = 0;
        }

        PrintTimer++;
        if (PrintTimer == PrintRate)
        {
            // Print information if needed
            PrintTimer = 0;
        }

        RunTime += Dt;
        if (TotalRunTime < RunTime)
        {
            Pause = 1;
        }
    }
}