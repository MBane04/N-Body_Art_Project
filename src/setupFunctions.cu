#include "hip/hip_runtime.h"
#include "./header.h"

/*
  This file contains the following functions:
        void setSimulationParameters();
        void allocateMemory();
        void setInitialConditions();
        void setup();

*/

void setSimulationParameters()
{
    if(PreviousRunToggle == 0) numBodies = 0; //start with no bodies, a blank canvas

    TotalRunTime = 10000.0;
    Dt = 0.002;

    // This is a lennard-Jones type force G*m1*m2/(r^2) - H*m1*m2/(r^4).
    // If you want a gravity type force just set G to your gravity and set H equal 0.
    G = 0.03;
    H = 0.00001;
    Epsilon = 0.01;
    MassOfBody = 1.0;
    DiameterOfBody = 0.2;
    VelocityMax = 10.0;
    Drag = 0.001;
    DrawRate = 8;
    PrintRate = 100;
}

void allocateMemory()
{
    // Allocate initial memory for the bodies array
    bodies = (Body*)malloc(capacity * sizeof(Body));
    if (bodies == NULL) 
    {
        fprintf(stderr, "Initial memory allocation failed\n");
        exit(1);
    }
    //printf("Initial memory allocated with capacity: %d\n", capacity);
}

void setInitialConditions()
{
    float dx, dy, dz, d, d2;
    int test;
    time_t t;
    
    srand((unsigned) time(&t));
    for(int i = 0; i < numBodies; i++)
    {
        bodies[i].id = i;
        test = 0;
        while(test == 0)
        {
            // Get random number between -1 at 1.
            bodies[i].pos.x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
            bodies[i].pos.y= ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
            bodies[i].pos.z= 0.0;  //((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
            test = 1;
            
            for(int j = 0; j < i; j++)
            {
                dx = bodies[i].pos.x - bodies[j].pos.x;
                dy = bodies[i].pos.y - bodies[j].pos.y;
                dz = bodies[i].pos.z - bodies[j].pos.z;
                d2  = dx*dx + dy*dy + dz*dz;
                d = sqrt(d2);
                if(d < DiameterOfBody)
                {
                    test = 0;
                    break;
                }
            }
            
            if(test == 1)
            {
                bodies[i].vel.x = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
                bodies[i].vel.y = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
                bodies[i].vel.z = 0.0;  //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
                
                bodies[i].color.x = ((float)rand()/(float)RAND_MAX);
                bodies[i].color.y = ((float)rand()/(float)RAND_MAX);
                bodies[i].color.z = ((float)rand()/(float)RAND_MAX);
            }
        }
        //set the radius of the body
        bodies[i].radius =((float)rand()/(float)RAND_MAX)* DiameterOfBody/2.0;

        //initialize everything else to zero
        bodies[i].force.x = 0.0;
        bodies[i].force.y = 0.0;
        bodies[i].force.z = 0.0;
        bodies[i].movement = 0;
        bodies[i].isSolid = true;
    }
}

void setup()
{
	allocateMemory();
    if (PreviousRunToggle == 1)
    {
        // Read the previous simulation parameters from the specified file
        readBodiesFromFile(PreviousRunFile.c_str());
        setSimulationParameters();
        //zeroOutSystem();
    }
    else
    {
        // Set up a new simulation
        setSimulationParameters();
        allocateMemory();
        setInitialConditions();
        //zeroOutSystem();
    }

    DrawTimer = 0;
    PrintRate = 0;
    RunTime = 0.0;
    Trace = 0;
    Pause = 1;
    MovieOn = 0;
    terminalPrint();
}