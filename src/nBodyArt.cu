#include "hip/hip_runtime.h"
// nvcc nBodyArtB.cu -o nBodyArt -lglut -lm -lGLU -lGL																																							
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <signal.h>
#include <cmath>
using namespace std;

FILE* ffmpeg;


// defines for terminal stuff.
#define BOLD_ON  "\e[1m"
#define BOLD_OFF   "\e[m"
#define INITIAL_CAPACITY 100

FILE* MovieFile;

// Globals
int NumberOfInitBodies;
float TotalRunTime;
float Dt;
float G;
float H;
float Epsilon;
float MassOfBody;
float DiameterOfBody;
float VelocityMax;
float Drag;
int DrawRate;
int PrintRate;

// Other Globals
int Pause;
//float *BodyPositionX, *BodyPositionY, *BodyPositionZ;
//float *BodyVelocityX, *BodyVelocityY, *BodyVelocityZ;
//float *BodyForceX, *BodyForceY, *BodyForceZ;
//float *BodyColorX, *BodyColorY, *BodyColorZ;
int DrawTimer, PrintTimer;
float RunTime;
int* Buffer;
int MovieOn;
int MovieFlag;
int Trace;
float MouseX, MouseY, MouseZ;
float newBodyRadius = 1.0;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize; 
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;



typedef struct
{
	int id;
	bool isSolid;
	float4 color;
	int movement; //preconfigured movement pattern
	float4 pos;
	float4 vel;
	float4 force;
	float radius;
} Body;

// Prototyping functions
void setSimulationParameters();
void allocateMemory();
void setInitialConditions();
void drawPicture();
void nBody();
void errorCheck(const char*);
void terminalPrint();
void setup();
void movieOn();
void movieOff();
void screenShot();
float4 centerOfMass();
float4 linearVelocity();
void zeroOutSystem();
void addBody(Body newBody);

//Toggles
int NewBodyToggle = 0; // 0 if not currently adding a new body, 1 if currently adding a new body.
bool isOrthogonal = true;
int PreviousRunToggle = 1; // do you want to run a previous simulation or start a new one?
string PreviousRunFile = "simulationparams"; // The file name of the previous simulation you want to run.
int ColorToggle = 0; //15 possible values
int HotkeyPrint = 0; // 0 if not currently printing hotkeys, 1 if currently printing hotkeys.
int NewBodyMovement = 0; // 0 if random movement, 1 if circular movement
bool NewBodySolid = true; // 0 if not solid, 1 if solid


typedef struct //stores colors for Starry night
{
    float4 paris_m;
    float4 manz;
    float4 outer_space;
    float4 curious_blue;
    float4 tahuna_sands;
    float4 livid_brown;
    float4 neptune;
    float4 lochmara;
    float4 regal_blue;
    float4 vis_vis;
    float4 light_curious_blue;
    float4 ironside_grey;
    float4 yellow;
    float4 deco;
    float4 astronaut_blue;
} Colors;

Colors colors = { // assigns values corresponding to the colors in the struct
    {49.0/255.0, 39.0/255.0, 96.0/255.0, 1.0},
    {228.0/255.0, 219.0/255.0, 85.0/255.0, 1.0},
    {65.0/255.0, 74.0/255.0, 76.0/255.0, 1.0},
    {21.18/255.0, 44.31/255.0, 77.65/255.0, 1.0},
    {93.0/255.0, 94.0/255.0, 78.0/255.0, 1.0},
    {49.0/255.0, 42.0/255.0, 41.0/255.0, 1.0},
    {49.0/255.0, 72.0/255.0, 73.0/255.0, 1.0},
    {50.0/255.0, 100.0/255.0, 150.0/255.0, 1.0},
    {14.0/255.0, 54.0/255.0, 87.0/255.0, 1.0},
    {249.0/255.0, 228.0/255.0, 150.0/255.0, 1.0},
    {15.0/255.0, 59.0/255.0, 82.0/255.0, 1.0},
    {40.0/255.0, 40.0/255.0, 38.0/255.0, 1.0},
    {244.0/255.0, 179.0/255.0, 5.0/255.0, 1.0},
    {198.0/255.0, 202.0/255.0, 116.0/255.0, 1.0},
    {42.0/255.0, 75.0/255.0, 124.0/255.0, 1.0}
};

float4 getColor(const char* colorName) { //to assign colors to the new body, call this function with the color name
    if (strcmp(colorName, "paris_m") == 0) return colors.paris_m;
    if (strcmp(colorName, "manz") == 0) return colors.manz;
    if (strcmp(colorName, "outer_space") == 0) return colors.outer_space;
    if (strcmp(colorName, "curious_blue") == 0) return colors.curious_blue;
    if (strcmp(colorName, "tahuna_sands") == 0) return colors.tahuna_sands;
    if (strcmp(colorName, "livid_brown") == 0) return colors.livid_brown;
    if (strcmp(colorName, "neptune") == 0) return colors.neptune;
    if (strcmp(colorName, "lochmara") == 0) return colors.lochmara;
    if (strcmp(colorName, "regal_blue") == 0) return colors.regal_blue;
    if (strcmp(colorName, "vis_vis") == 0) return colors.vis_vis;
    if (strcmp(colorName, "light_curious_blue") == 0) return colors.light_curious_blue;
    if (strcmp(colorName, "ironside_grey") == 0) return colors.ironside_grey;
    if (strcmp(colorName, "yellow") == 0) return colors.yellow;
    if (strcmp(colorName, "deco") == 0) return colors.deco;
    if (strcmp(colorName, "astronaut_blue") == 0) return colors.astronaut_blue;
    return (float4){0.0, 0.0, 0.0, 1.0}; // Default value
}

Body* bodies = NULL;
int numBodies = NumberOfInitBodies;
int capacity = INITIAL_CAPACITY; // Initial capacity of the bodies array


void readBodiesFromFile(const char* filename)
{
    FILE* file = fopen(filename, "r");
    if (file == NULL)
    {
        fprintf(stderr, "Error: Could not open file %s for reading\n", filename);
        return;
    }

    // Read the number of bodies from the top of the file
    int numBodiesFromFile;
    if (fscanf(file, "Number of bodies: %d\n", &numBodiesFromFile) != 1)
    {
        fprintf(stderr, "Error: Could not read the number of bodies from the file\n");
        fclose(file);
        return;
    }

    // Allocate memory based on the number of bodies
    capacity = numBodiesFromFile;
    bodies = (Body*)malloc(capacity * sizeof(Body));
    if (bodies == NULL)
    {
        fprintf(stderr, "Memory allocation failed\n");
        fclose(file);
        exit(1);
    }
    printf("Initial memory allocated with capacity: %d\n", capacity);

    // Skip the header line
    char header[256];
    fgets(header, sizeof(header), file);

    // Read body information
    for (int i = 0; i < numBodiesFromFile; i++)
    {
        Body newBody;
        int isSolid;
        float color_x, color_y, color_z, color_w;
        float pos_x, pos_y, pos_z;
        float vel_x, vel_y, vel_z;
        float force_x, force_y, force_z;
        int result = fscanf(file, "%d, %d, (%f, %f, %f, %f), %d, (%f, %f, %f), (%f, %f, %f), (%f, %f, %f), %f\n",
                            &newBody.id,
                            &isSolid,
                            &color_x, &color_y, &color_z, &color_w,
                            &newBody.movement,
                            &pos_x, &pos_y, &pos_z,
                            &vel_x, &vel_y, &vel_z,
                            &force_x, &force_y, &force_z,
                            &newBody.radius);
        newBody.isSolid = (bool)isSolid;
        newBody.color = make_float4(color_x, color_y, color_z, color_w);
        newBody.pos = make_float4(pos_x, pos_y, pos_z, 1.0f);
        newBody.vel = make_float4(vel_x, vel_y, vel_z, 0.0f);
        newBody.force = make_float4(force_x, force_y, force_z, 0.0f);

        if (result == 17)
        {
            addBody(newBody);
            printf("Read body %d: id=%d, isSolid=%d, color=(%f, %f, %f, %f), movement=%d, pos=(%f, %f, %f), vel=(%f, %f, %f), force=(%f, %f, %f), radius=%f\n",
                   i, newBody.id, newBody.isSolid, newBody.color.x, newBody.color.y, newBody.color.z, newBody.color.w,
                   newBody.movement, newBody.pos.x, newBody.pos.y, newBody.pos.z,
                   newBody.vel.x, newBody.vel.y, newBody.vel.z,
                   newBody.force.x, newBody.force.y, newBody.force.z,
                   newBody.radius);
        }
        else
        {
            fprintf(stderr, "Error: fscanf read %d values instead of 17\n", result);
            break;
        }
    }

    fclose(file);
    printf("Body information read from %s\n", filename);

    // Update numBodies
    numBodies = numBodiesFromFile;
}

void writeBodiesToFile(const char* filename)
{
    FILE* file = fopen(filename, "w");
    if (file == NULL)
    {
        fprintf(stderr, "Error: Could not open file %s for writing\n", filename);
        return;
    }

	// Write the number of bodies at the end of the file
    fprintf(file, "Number of bodies: %d\n", numBodies);


    fprintf(file, "ID, IsSolid, Color (R, G, B, A), Movement, Position (X, Y, Z), Velocity (X, Y, Z), Force (X, Y, Z), Radius\n");
    for (int i = 0; i < numBodies; i++)
    {
        fprintf(file, "%d, %d, (%f, %f, %f, %f), %d, (%f, %f, %f), (%f, %f, %f), (%f, %f, %f), %f\n",
                bodies[i].id,
                bodies[i].isSolid,
                bodies[i].color.x, bodies[i].color.y, bodies[i].color.z, bodies[i].color.w,
                bodies[i].movement,
                bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z,
                bodies[i].vel.x, bodies[i].vel.y, bodies[i].vel.z,
                bodies[i].force.x, bodies[i].force.y, bodies[i].force.z,
                bodies[i].radius);
    }



    fclose(file);
    printf("Body information written to %s\n", filename);
}

void addBody(Body newBody) 
{
    // Reallocate memory to accommodate the new body
	
    if (numBodies >= capacity) //if the new body will exceed the current capacity
	{
        capacity *= 2; //double the capacity
        Body* temp = (Body*)realloc(bodies, capacity * sizeof(Body)); //reallocate memory to accommodate the new body
        if (temp == NULL)  //if memory allocation fails
		{
            fprintf(stderr, "Memory allocation failed\n");
            exit(1);
        }
        bodies = temp;//assign the new memory to the bodies array, so long as memory allocation was successful
		//printf("Reallocated memory to capacity: %d\n", capacity);
    }


	//
	if(newBody.movement == 0) //random movement
	{
		newBody.vel.x = ((float)rand()/(float)RAND_MAX)*2.0f - 1.0f;
		newBody.vel.y = ((float)rand()/(float)RAND_MAX)*2.0f - 1.0f;
		newBody.vel.z = 0.0;
	}
	if (newBody.movement == 1) //still
	{
        newBody.vel.x = 0.0f;
        newBody.vel.y = 0.0f;
        newBody.vel.z = 0.0f;
	}


    /// Add the new body to the array
	bodies[numBodies] = newBody;

    // Increment the number of bodies
    numBodies++;

	//for debugging
	//printf("Body %d added at (%f, %f, %f) with velocity (%f, %f, %f)\n", newBody.id, newBody.pos.x, newBody.pos.y, newBody.pos.z, newBody.vel.x, newBody.vel.y, newBody.vel.z);
}

void freeBodies() 
{
    free(bodies);
}

void setup()
{
	allocateMemory();
    if (PreviousRunToggle == 1)
    {
        // Read the previous simulation parameters from the specified file
        readBodiesFromFile(PreviousRunFile.c_str());
        setSimulationParameters();
        zeroOutSystem();
    }
    else
    {
        // Set up a new simulation
        setSimulationParameters();
        allocateMemory();
        setInitialConditions();
        zeroOutSystem();
    }

    DrawTimer = 0;
    PrintRate = 0;
    RunTime = 0.0;
    Trace = 0;
    Pause = 1;
    MovieOn = 0;
    terminalPrint();
}

void Display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	drawPicture();
	glutSwapBuffers();
}

void idle()
{
    if (NewBodyToggle == 1)
    {
        drawPicture();
    }
    else
    {
        nBody();
    }
}

void reshape(int w, int h)
{
    // Prevent division by zero
    if (h == 0) h = 1;

    // Calculate the aspect ratio of the window
    float aspectRatio = (float)w / (float)h; //currently 3000/1500 = 2

    // Set the viewport to cover the new window
    glViewport(0, 0, (GLsizei)w, (GLsizei)h);

    // Set the projection matrix
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    // Adjust the projection matrix to maintain the aspect ratio of the bodies
    if (isOrthogonal) 
	{
        if (aspectRatio >= 1.0f) 
		{
            // Window is wider than it is tall
            glOrtho(-1.0 * aspectRatio, 1.0 * aspectRatio, -1.0, 1.0, Near, Far);
        } 
		else 
		{
            // Window is taller than it is wide
            glOrtho(-1.0, 1.0, -1.0 / aspectRatio, 1.0 / aspectRatio, Near, Far);
        }
    } 
	else 
	{
        if (aspectRatio >= 1.0f) 
		{
            // Window is wider than it is tall
            glFrustum(-0.2 * aspectRatio, 0.2 * aspectRatio, -0.2, 0.2, Near, Far);
        } else 
		{
            // Window is taller than it is wide
            glFrustum(-0.2, 0.2, -0.2 / aspectRatio, 0.2 / aspectRatio, Near, Far);
        }
    }

    // Switch back to the modelview matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
}

void KeyPressed(unsigned char key, int x, int y)
{	
	if(key == 'q')
	{
		// Check if ffmpeg is not NULL before closing
        if (ffmpeg != NULL) 
		{
            pclose(ffmpeg);
            ffmpeg = NULL; // Optionally set to NULL after closing
        } 
		else 
		{
            fprintf(stderr, "Warning: Attempted to close a NULL file pointer\n");
        }
        glutDestroyWindow(Window);
        printf("\nw Good Bye\n");
        exit(0);
	}
	// if(key == 'v') //not much need for this anymore
   	// {
    //     // Toggle the view mode
    //     isOrthogonal = !isOrthogonal;

    //     // Call reshape to update the projection matrix
    //     reshape(glutGet(GLUT_WINDOW_WIDTH), glutGet(GLUT_WINDOW_HEIGHT));

    //     // Redraw the scene
    //     glutPostRedisplay();
   	//  }
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 't') // Turns tracers on and off
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 'M')  // Movie on/off
	{
		if(MovieFlag == 0) 
		{
			MovieFlag = 1;
			movieOn();
		}
		else 
		{
			MovieFlag = 0;
			movieOff();
		}
		terminalPrint();
	}
	
	if(key == 'S')  // Screenshot
	{	
		screenShot();
		terminalPrint();
	}
	if (key == 'n') // Add a new body
	{
		if(NewBodyToggle == 0) NewBodyToggle = 1;
		else NewBodyToggle = 0;
		terminalPrint();
	}
	if(key == ']')  
	{
		newBodyRadius += 0.01;
		terminalPrint();
		//printf("\n Your selection area = %f times the radius of atrium. \n", HitMultiplier);
	}
	if(key == '[')
	{
		newBodyRadius -= 0.01;
		if(newBodyRadius < 0.0) newBodyRadius = 0.0;
		terminalPrint();
		//printf("\n Your selection area = %f times the radius of atrium. \n", HitMultiplier);
	}
	if(key == 's')
	{
		writeBodiesToFile("simulationparams");
	}

    if(NewBodyToggle == 1)
    {
        if (key == 'l') // cycle through colors, forward
        {
            if (ColorToggle < 15)
            {
                ColorToggle++;
            }
            else
            {
                ColorToggle = 1;
            }
            terminalPrint();
        }
        if (key == 'k') // cycle through colors, backward
        {
            if (ColorToggle > 1)
            {
                ColorToggle--;
            }
            else
            {
                ColorToggle = 15;
            }
            terminalPrint();
        }
        //set movement pattern
        //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!CHANGES NEED TO BE MADE LATER!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        if (key == 'm')
        {
            printf("Enter the movement pattern for the new body: ");
            scanf("%d", &NewBodyMovement);
            if (NewBodyMovement != 0 && NewBodyMovement != 1) //change this when we actually make new bodies
            {
                printf("Invalid movement pattern.\n");
                NewBodyMovement = 0;
            }
            terminalPrint();
        }

        if(key == 'i')//is the new body solid?
        {
            if(NewBodySolid == true)
            {
                NewBodySolid = false;
            }
            else
            {
                NewBodySolid = true;
            }
            terminalPrint();
        }
       
    }
}

void mousePassiveMotionCallback(int x, int y) 
{

	// Convert window coordinates to OpenGL coordinates
		MouseX = ( 2.0*x/XWindowSize - 1.0) *2.8 + 1.0;
		MouseY = (-2.0*y/YWindowSize + 1.0)*1.5 - 0.5;

    // Print the converted coordinates for debugging
    //printf("MouseX: %f, MouseY: %f\n", MouseX, MouseY);


    // Redraw the scene
    //glutPostRedisplay();
}

// This is called when you push a mouse button.
void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{	
		if(button == GLUT_LEFT_BUTTON)
		{	
			if(NewBodyToggle == 1)
			{

                //generate random numbers for all the properties of the new body
				
                int index = numBodies; // Define and initialize index

				// Convert window coordinates to OpenGL coordinates
				MouseX = ( 2.0*x/XWindowSize - 1.0) *2.8 + 1.0;
				MouseY = (-2.0*y/YWindowSize + 1.0)*1.5 - 0.5;
                MouseZ = 0.0f;

                // Print the converted coordinates for debugging
                printf("MouseX: %f, MouseY: %f, MouseZ: %f\n", MouseX, MouseY, MouseZ);

                Body newBody; //create a new body with the body struct

                // Set the color of the new body based on the ColorToggle

                if(ColorToggle == 1)
                {
                    newBody.color = getColor("paris_m");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 2)
                {
                    newBody.color = getColor("manz");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 3)
                {
                    newBody.color = getColor("outer_space");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 4)
                {
                    newBody.color = getColor("curious_blue");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 5)
                {
                    newBody.color = getColor("tahuna_sands");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 6)
                {
                    newBody.color = getColor("livid_brown");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 7)
                {
                    newBody.color = getColor("neptune");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 8)
                {
                    newBody.color = getColor("lochmara");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 9)
                {
                    newBody.color = getColor("regal_blue");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 10)
                {
                    newBody.color = getColor("vis_vis");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 11)
                {
                    newBody.color = getColor("light_curious_blue");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 12)
                {
                    newBody.color = getColor("ironside_grey");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 13)
                {
                    newBody.color = getColor("yellow");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 14)
                {
                    newBody.color = getColor("deco");
                    HotkeyPrint = 0;
                }
                else if(ColorToggle == 15)
                {
                    newBody.color = getColor("astronaut_blue");
                    HotkeyPrint = 0;
                }
                else
                {
                    newBody.color =  {1.0f, 1.0f, 1.0f, 1.0f}; //default
                }

                //assign all the properties of the new body
                newBody.id = index;
                newBody.isSolid = true;
                newBody.movement = NewBodyMovement;
                newBody.pos = {MouseX, MouseY, MouseZ, 1.0f}; // Directly assign values to float4
                newBody.force = {0.0f, 0.0f, 0.0f, 0.0f}; // Directly assign values to float4
				newBody.radius = newBodyRadius*DiameterOfBody/2.0;

                addBody(newBody);
			}
		}
		else if(button == GLUT_RIGHT_BUTTON) // Right Mouse button down
		{
			// Do stuff in here if you choose to when the right mouse button is pressed.
		}
		else if(button == GLUT_MIDDLE_BUTTON)
		{
			// Do stuff in here if you choose to when the middle mouse button is pressed.
		}
	}
	
	// If no mouse button is down (state 0, they don't have a nice word like GLUT_NOT_DOWN) 
	// but you move the mouse wheel this is called.
	if(state == 0)
	{
		// When you turn the mouse whell forward this is called.
		if(button == 3)
		{
			EyeZ -=0.1;
			
		}
		
		// When you turn the mouse whell backward this is called.
		else if(button == 4)
		{
			EyeZ += 0.1;
			
		}
	}
	glLoadIdentity();
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	//glutPostRedisplay();
}


string getTimeStamp()
{
	// Want to get a time stamp string representing current date/time, so we have a
	// unique name for each video/screenshot taken.
	time_t t = time(0); 
	struct tm * now = localtime( & t );
	int month = now->tm_mon + 1, day = now->tm_mday, year = now->tm_year, 
				curTimeHour = now->tm_hour, curTimeMin = now->tm_min, curTimeSec = now->tm_sec;
	stringstream smonth, sday, syear, stimeHour, stimeMin, stimeSec;
	smonth << month;
	sday << day;
	syear << (year + 1900); // The computer starts counting from the year 1900, so 1900 is year 0. So we fix that.
	stimeHour << curTimeHour;
	stimeMin << curTimeMin;
	stimeSec << curTimeSec;
	string timeStamp;
	if (curTimeMin <= 9)	
		timeStamp = smonth.str() + "-" + sday.str() + "-" + syear.str() + '_' + stimeHour.str() + ".0" + stimeMin.str() + 
					"." + stimeSec.str();
	else			
		timeStamp = smonth.str() + "-" + sday.str() + '-' + syear.str() + "_" + stimeHour.str() + "." + stimeMin.str() +
					"." + stimeSec.str();
	return timeStamp;
}

// Signal handler for SIGPIPE
void handle_sigpipe(int sig)
{
    fprintf(stderr, "Caught SIGPIPE signal: %d\n", sig);
}

void movieOn()
{
    // Register the SIGPIPE signal handler
    signal(SIGPIPE, handle_sigpipe);

    string ts = getTimeStamp();
    ts.append(".mp4");

    // Convert the x and y window size to a string of format "XsizexYsize"
    stringstream ss;
    ss << XWindowSize << "x" << YWindowSize;
    string windowSize = ss.str();

    // Setting up the movie buffer with the dynamic window size
    string baseCommand = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s " + windowSize + " -i - "
                         "-c:v libx264rgb -threads 0 -preset fast -y -pix_fmt yuv420p -crf 0 -vf vflip 2>ffmpeg_error.log ";

    string z = baseCommand + ts;

    const char *ccx = z.c_str();
    MovieFile = popen(ccx, "w");

    // Check if popen was successful
    if (MovieFile == NULL) {
        fprintf(stderr, "Error: Failed to open movie file with popen\n");
        return;
    }

    // Allocate buffer
    Buffer = (int*)malloc(XWindowSize * YWindowSize * sizeof(int));

    // Check if malloc was successful
    if (Buffer == NULL) {
        fprintf(stderr, "Error: Failed to allocate memory for buffer\n");
        pclose(MovieFile);
        MovieFile = NULL;
        return;
    }

    MovieOn = 1;
    printf("Movie recording started successfully\n");
}

void movieOff()
{
	if(MovieOn == 1) 
	{
		pclose(MovieFile);
	}
	free(Buffer);
	MovieOn = 0;
}

void screenShot()
{	
	int pauseFlag;
	FILE* ScreenShotFile;
	int* buffer;

	//convert the x and y windowsize to a string of format "XsizexYsize"
    stringstream ss;
    ss << XWindowSize << "x" << YWindowSize;
    string windowSize = ss.str();

    // Construct the ffmpeg command with the dynamic window size
    string baseCommand = "ffmpeg -loglevel quiet -framerate 60 -f rawvideo -pix_fmt rgba -s " + windowSize + " -i - "
                         "-c:v libx264rgb -threads 0 -preset fast -y -crf 0 -vf vflip output1.mp4";
    const char* cmd = baseCommand.c_str();
	//const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
	//              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
	ScreenShotFile = popen(cmd, "w");
	buffer = (int*)malloc(XWindowSize*YWindowSize*sizeof(int));
	
	if(Pause == 0) 
	{
		Pause = 1;
		pauseFlag = 0;
	}
	else
	{
		pauseFlag = 1;
	}
	
	for(int i =0; i < 1; i++)
	{
		drawPicture();
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
		fwrite(buffer, sizeof(int)*XWindowSize*YWindowSize, 1, ScreenShotFile);
	}
	
	pclose(ScreenShotFile);
	free(buffer);

	string ts = getTimeStamp(); // Only storing in a separate variable for debugging purposes.
	string s = "ffmpeg -loglevel quiet -i output1.mp4 -qscale:v 1 -qmin 1 -qmax 1 " + ts + ".jpeg";
	// Convert back to a C-style string.
	const char *ccx = s.c_str();
	system(ccx);
	system("rm output1.mp4");
	printf("\nScreenshot Captured: \n");
	cout << "Saved as " << ts << ".jpeg" << endl;
	
	Pause = pauseFlag;
	//ffmpeg -i output1.mp4 output_%03d.jpeg
}

void setSimulationParameters()
{
	//numBodies = 16;

	TotalRunTime = 10000.0;

	Dt = 0.002;

	// This is a lennard-Jones type force G*m1*m2/(r^2) - H*m1*m2/(r^4).
	// If you want a gravity type force just set G to your gravity and set H equal 0.
	G = 0.03;

	H = 0.00001;

	Epsilon = 0.01;

	MassOfBody = 1.0;

	DiameterOfBody = 0.2;

	VelocityMax = 10.0;

	Drag = 0.001;

	DrawRate = 8;
	
	PrintRate = 100;
}

void allocateMemory()
{
// Allocate initial memory for the bodies array
    bodies = (Body*)malloc(capacity * sizeof(Body));
    if (bodies == NULL) 
    {
        fprintf(stderr, "Initial memory allocation failed\n");
        exit(1);
    }
    printf("Initial memory allocated with capacity: %d\n", capacity);
}

void setInitialConditions()
{
    float dx, dy, dz, d, d2;
    int test;
	time_t t;
	
	srand((unsigned) time(&t));
	for(int i = 0; i < numBodies; i++)
	{
		bodies[i].id = i;
		test = 0;
		while(test == 0)
		{
			// Get random number between -1 at 1.
			bodies[i].pos.x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			bodies[i].pos.y= ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			bodies[i].pos.z= 0.0;  //((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			test = 1;
			
			for(int j = 0; j < i; j++)
			{
				dx = bodies[i].pos.x - bodies[j].pos.x;
				dy = bodies[i].pos.y - bodies[j].pos.y;
				dz = bodies[i].pos.z - bodies[j].pos.z;
				d2  = dx*dx + dy*dy + dz*dz;
				d = sqrt(d2);
				if(d < DiameterOfBody)
				{
					test = 0;
					break;
				}
			}
			
			if(test == 1)
			{
				bodies[i].vel.x = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				bodies[i].vel.y = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				bodies[i].vel.z = 0.0;  //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				
				bodies[i].color.x = ((float)rand()/(float)RAND_MAX);
				bodies[i].color.y = ((float)rand()/(float)RAND_MAX);
				bodies[i].color.z = ((float)rand()/(float)RAND_MAX);
			}
		}
		//set the radius of the body
		bodies[i].radius =((float)rand()/(float)RAND_MAX)* DiameterOfBody/2.0;

		//initialize everything else to zero
		bodies[i].force.x = 0.0;
		bodies[i].force.y = 0.0;
		bodies[i].force.z = 0.0;
		bodies[i].movement = 0;
		bodies[i].isSolid = true;
		
	}
}

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < numBodies; i++)
	{
    	centerOfMass.x += bodies[i].pos.x*MassOfBody;
		centerOfMass.y += bodies[i].pos.y*MassOfBody;
		centerOfMass.z += bodies[i].pos.z*MassOfBody;
		totalMass += MassOfBody;
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float totalMass;
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < numBodies; i++)
	{
    	linearVelocity.x += bodies[i].vel.x*MassOfBody;
		linearVelocity.y += bodies[i].vel.y*MassOfBody;
		linearVelocity.z += bodies[i].vel.z*MassOfBody;
		totalMass += MassOfBody;
	}
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	
	return(linearVelocity);
}

void zeroOutSystem()
{
	float4 pos, vel;
	pos = centerOfMass();
	vel = linearVelocity();
		
	for(int i = 0; i < numBodies; i++)
	{
		bodies[i].pos.x -= pos.x;
		bodies[i].pos.y -= pos.y;
		bodies[i].pos.z -= pos.z;
		
		bodies[i].vel.x -= vel.x;
		bodies[i].vel.y -= vel.y;
		bodies[i].vel.z -= vel.z;
	}
}

void drawPicture()
{


    if (Trace == 0)
    {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    }

    if (NewBodyToggle == 1)
    {
        float4 mouseColor;
        // Set mouse to look like a new body

        // float4 paris_m = (49.0/255.0, 39.0/255.0, 96.0/255.0);
        // float4 manz =(228.0/255.0, 219.0/255.0, 85.0/255.0);
        // float4 outer_space =(65.0/255.0, 74.0/255.0, 76.0/255.0);
        // float4 curious_blue = (21.18/255.0, 44.31/255.0, 77.65/255.0);
        // float4 tahuna_sands = (93.0/255.0, 94.0/255.0, 78.0/255.0);
        // float4 livid_brown = (49.0/255.0, 42.0/255.0, 41.0/255.0);
        // float4 neptune = (49.0/255.0, 72.0/255.0, 73.0/255.0);
        // float4 lochmara = (50.0/255.0, 100.0/255.0, 150.0/255.0);
        // float4 regal_blue = (14.0/255.0, 54.0/255.0, 87.0/255.0);
        // float4 vis_vis = (249.0/255.0, 228.0/255.0, 150.0/255.0);
        // float4 light_curious_blue = (15.0/255.0, 59.0/255.0, 82.0/255.0);
        // float4 ironside_grey = (40.0/255.0, 40.0/255.0, 38.0/255.0);
        // float4 yellow = (244.0/255.0, 179.0/255.0, 5.0/255.0);
        // float4 deco = (198.0/255.0, 202.0/255.0, 116.0/255.0);
        // float4 astronaut_blue = (42.0/255.0, 75.0/255.0, 124.0/255.0);

        if (ColorToggle == 1)
        {
            //color paris m
            mouseColor = getColor("paris_m");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 2)
        {
            //color manz
            mouseColor = getColor("manz");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 3)
        {
            //color outer space
            mouseColor = getColor("outer_space");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 4)
        {
            //color curious blue
            mouseColor = getColor("curious_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 5)
        {
            //color tahuna sands
            mouseColor = getColor("tahuna_sands");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 6)
        {
            //color livid brown
            mouseColor = getColor("livid_brown");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 7)
        {
            //color neptune
            mouseColor = getColor("neptune");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 8)
        {
            //color lochmara
            mouseColor = getColor("lochmara");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 9)
        {
            //color regal blue
            mouseColor = getColor("regal_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 10)
        {
            //color vis vis
            mouseColor = getColor("vis_vis");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 11)
        {
            //color light curious blue
            mouseColor = getColor("light_curious_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 12)
        {
            //color ironside grey
            mouseColor = getColor("ironside_grey");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 13)
        {
            //color yellow
            mouseColor = getColor("yellow");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 14)
        {
            //color deco
            mouseColor = getColor("deco");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 15)
        {
            //color astronaut blue
            mouseColor = getColor("astronaut_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else
        {
            //color white
            glColor3d(1.0, 1.0, 1.0);
        }
        glPushMatrix();
        glTranslatef(MouseX, MouseY, MouseZ);
        glutSolidSphere(newBodyRadius * DiameterOfBody / 2.0, 20, 20);
        glPopMatrix();
    }

    for (int i = 0; i < numBodies; i++)
    {
        glColor3d(bodies[i].color.x, bodies[i].color.y, bodies[i].color.z);
        glPushMatrix();
        glTranslatef(bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z);
        glutSolidSphere(bodies[i].radius, 20, 20);
        glPopMatrix();
    }

    glutSwapBuffers();

    if (MovieOn == 1)
    {
        glReadPixels(0, 0, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
        fwrite(Buffer, sizeof(int) * XWindowSize * YWindowSize, 1, MovieFile);
    }
}

void getForces(Body* bodies, float mass, float G, float H, float Epsilon, float drag, float dt, int n)
{
	float dx, dy, dz, d2, d;
    float forceMag;
    float inOut;
	float kSphereReduction = 0.5;
	float dvx, dvy, dvz;
	float kSphere = 10000;

    // Initialize forces to zero
    for (int i = 0; i < n; i++)
    {
        bodies[i].force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }

    // Calculate forces
    for (int i = 0; i < n; i++)
    {
        for (int j = i + 1; j < n; j++)
        {
            dx = bodies[j].pos.x - bodies[i].pos.x;
            dy = bodies[j].pos.y - bodies[i].pos.y;
            dz = bodies[j].pos.z - bodies[i].pos.z;
            d2 = dx * dx + dy * dy + dz * dz + Epsilon;
            d = sqrt(d2);
			if (d < 1e-6) 
			{
                fprintf(stderr, "Warning: Small distance in force calculation, skipping\n");
                continue;
            }
            //forceMag = (G * mass * mass) / d2 - (H * mass * mass) / (d2 * d2); // gravitational force
            forceMag = 0.0; //No force between bodies. Each body acts individually.

            float3 force = make_float3(forceMag * dx / d,
                                       forceMag * dy / d,
                                       forceMag * dz / d);

            if(bodies[i].isSolid ^ bodies[j].isSolid) //bitwise XOR. If one is solid and the other is not, and only then, do the following.
            {
                float combinedDiamter = bodies[i].radius + bodies[j].radius;
                if(d < combinedDiamter) //if the balls touch. i.e if the distance betweeen < both radii
			    {
                    
                    dvx = bodies[j].vel.x - bodies[i].vel.x;
                    dvy = bodies[j].vel.y - bodies[i].vel.y;
                    dvz = bodies[j].vel.z - bodies[i].vel.z;
                    inOut = dx*dvx + dy*dvy + dz*dvz;
                    if(inOut < 0.0) forceMag = kSphere*(combinedDiamter - d); // If inOut is negative the sphere are converging.
                    else forceMag = kSphereReduction*kSphere*(combinedDiamter - d); // If inOut is positive the sphere are diverging.
                    
                    // Doling out the force in the proper perfortions using unit vectors.
                    bodies[i].force.x -= forceMag*(dx/d);
                    bodies[i].force.y -= forceMag*(dy/d);
                    bodies[i].force.y -= forceMag*(dz/d);
                    // A force on me causes the opposite force on you. 
                    bodies[j].force.x += forceMag*(dx/d);
                    bodies[j].force.y += forceMag*(dy/d);
                    bodies[j].force.z += forceMag*(dz/d);
                }
            }

            bodies[i].force.x += force.x;
            bodies[i].force.y += force.y;
            bodies[i].force.z += force.z;

            bodies[j].force.x -= force.x;
            bodies[j].force.y -= force.y;
            bodies[j].force.z -= force.z;
        }
    }

	// Check for nan values
	for (int i = 0; i < n; i++) {
		if (isnan(bodies[i].color.x) || isnan(bodies[i].color.y) || isnan(bodies[i].color.z) || isnan(bodies[i].color.w) ||
			isnan(bodies[i].pos.x) || isnan(bodies[i].pos.y) || isnan(bodies[i].pos.z) ||
			isnan(bodies[i].vel.x) || isnan(bodies[i].vel.y) || isnan(bodies[i].vel.z) ||
			isnan(bodies[i].force.x) || isnan(bodies[i].force.y) || isnan(bodies[i].force.z) ||
			isnan(bodies[i].radius))
		{
			fprintf(stderr, "Error: Encountered nan value in body %d\n", bodies[i].id);
			exit(1);
		}
	}
	
}

void nBody()
{
    if (Pause != 1)
    {
        // Print initial positions and velocities
        // for (int i = 0; i < numBodies; i++)
        // {
        //     printf("Initial Body %d: pos=(%f, %f, %f), vel=(%f, %f, %f), force=(%f, %f, %f)\n",
        //            i, bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z,
        //            bodies[i].vel.x, bodies[i].vel.y, bodies[i].vel.z,
        //            bodies[i].force.x, bodies[i].force.y, bodies[i].force.z);
        // }

        // Calculate forces
        getForces(bodies, MassOfBody, G, H, Epsilon, Drag, Dt, numBodies);

        // Print positions, velocities, and forces after force calculation
        // for (int i = 0; i < numBodies; i++)
        // {
        //     printf("After Force Calculation Body %d: pos=(%f, %f, %f), vel=(%f, %f, %f), force=(%f, %f, %f)\n",
        //            i, bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z,
        //            bodies[i].vel.x, bodies[i].vel.y, bodies[i].vel.z,
        //            bodies[i].force.x, bodies[i].force.y, bodies[i].force.z);
        // }

        // Update positions and velocities
        for (int i = 0; i < numBodies; i++)
        {
            bodies[i].vel.x += ((bodies[i].force.x - Drag * bodies[i].vel.x) / MassOfBody) * Dt;
            bodies[i].vel.y += ((bodies[i].force.y - Drag * bodies[i].vel.y) / MassOfBody) * Dt;
            bodies[i].vel.z += ((bodies[i].force.z - Drag * bodies[i].vel.z) / MassOfBody) * Dt;

            bodies[i].pos.x += bodies[i].vel.x * Dt;
            bodies[i].pos.y += bodies[i].vel.y * Dt;
            bodies[i].pos.z += bodies[i].vel.z * Dt;

            // Check for nan values
            if (isnan(bodies[i].pos.x) || isnan(bodies[i].pos.y) || isnan(bodies[i].pos.z) ||
                isnan(bodies[i].vel.x) || isnan(bodies[i].vel.y) || isnan(bodies[i].vel.z) ||
                isnan(bodies[i].force.x) || isnan(bodies[i].force.y) || isnan(bodies[i].force.z))
            {
                fprintf(stderr, "Error: Encountered nan value in body %d during update\n", bodies[i].id);
                // exit(1);
            }
        }

        // Print positions and velocities after update
        // for (int i = 0; i < numBodies; i++)
        // {
        //     printf("After Update Body %d: pos=(%f, %f, %f), vel=(%f, %f, %f), force=(%f, %f, %f)\n",
        //            i, bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z,
        //            bodies[i].vel.x, bodies[i].vel.y, bodies[i].vel.z,
        //            bodies[i].force.x, bodies[i].force.y, bodies[i].force.z);
        // }

        DrawTimer++;
        if (DrawTimer == DrawRate)
        {
            drawPicture();
            DrawTimer = 0;
        }

        PrintTimer++;
        if (PrintTimer == PrintRate)
        {
            terminalPrint();
            PrintTimer = 0;
        }

        RunTime += Dt;
        if (TotalRunTime < RunTime)
        {
            printf("\n\n Done\n");
            exit(0);
        }
    }
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
    printf("\n");
	printf("\n S: Screenshot");
	
	printf("\n");
	printf("\n q: Terminates the simulation");

    printf("\n");
    printf("\n s: Save this run");

	printf("\n\n");
	printf("\033[0m");
	printf(" p: Pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf("\n");
	printf("\033[0m");
	printf(" t: Trace on/off toggle --> ");
	printf(" Trace is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;31mOn\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mOff\n" "\e[m");
	}
	//printf("\n");
	//printf("\033[0m");
	//printf(" v: Toggle view (Perspective/Orthogonal) --> ");
	//printf(" Current View: ");
	// if (isOrthogonal) 
	// {
	// 	printf("\e[1m" " \033[0;32mOrthogonal\n" "\e[m");
	// }
	// else 
	// {
	// 	printf("\e[1m" " \033[0;31mDefault\n" "\e[m");
	// }
	printf("\n M: Video On/Off toggle --> ");
	if (MovieFlag == 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "Video Recording Off\n" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Video Recording On\n" BOLD_OFF);
	}
	printf("\n n: Simulaton Mode Add View/Add Body Toggle --> Mode:");
	if (NewBodyToggle== 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "View" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Add Body" BOLD_OFF);
	}
	//controls for body placement
    if(NewBodyToggle == 1)
    {
        printf("\n");
        printf("\033[0m");
        printf(" [/]: Change radius of new body backwards/forwards\n");

        printf("\n");
        printf("\033[0m");
        printf(" k/l: Change color of new body backwards/forwards\n");
        printf(" Current Color: ");
        if (ColorToggle == 1)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Paris M" BOLD_OFF);
        }
        else if (ColorToggle == 2)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Manz" BOLD_OFF);
        }
        else if (ColorToggle == 3)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Outer Space" BOLD_OFF);
        }
        else if (ColorToggle == 4)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Curious Blue" BOLD_OFF);
        }
        else if (ColorToggle == 5)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Tahuna Sands" BOLD_OFF);
        }
        else if (ColorToggle == 6)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Livid Brown" BOLD_OFF);
        }
        else if (ColorToggle == 7)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Neptune" BOLD_OFF);
        }
        else if (ColorToggle == 8)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Lochmara" BOLD_OFF);
        }
        else if (ColorToggle == 9)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Regal Blue" BOLD_OFF);
        }
        else if (ColorToggle == 10)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Vis Vis" BOLD_OFF);
        }
        else if (ColorToggle == 11)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Light Curious Blue" BOLD_OFF);
        }
        else if (ColorToggle == 12)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Ironside Grey" BOLD_OFF);
        }
        else if (ColorToggle == 13)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Yellow" BOLD_OFF);
        }
        else if (ColorToggle == 14)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Deco" BOLD_OFF);
        }
        else if (ColorToggle == 15)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Astronaut Blue" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "DEFAULT" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf("m : set movement preset --> Current Preset:");
        if(NewBodyMovement == 0)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Random" BOLD_OFF);
        }
        else if(NewBodyMovement == 1)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Circle" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "DEFAULT" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf(" i: Body Solidity On/Off Toggle --> ");
        if (!NewBodySolid)
        {
            printf("\033[0;31m");
            printf(BOLD_ON "Solid Off" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Solid On" BOLD_OFF);
        }


    }
    printf("\n");
}




int main(int argc, char** argv)
{
    setup();

    XWindowSize = 3000;
    YWindowSize = 1500;

    // Clip planes
    Near = 0.2;
    Far = 30.0;

    // Direction here your eye is located location
    EyeX = 0.0;
    EyeY = 0.0;
    EyeZ = 2.0;

    // Where you are looking
    CenterX = 0.0;
    CenterY = 0.0;
    CenterZ = 0.0;

    // Up vector for viewing
    UpX = 0.0;
    UpY = 1.0;
    UpZ = 0.0;

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
    glutInitWindowSize(XWindowSize, YWindowSize);
    glutInitWindowPosition(5, 5);
    Window = glutCreateWindow("N Body");

    gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
    glMatrixMode(GL_MODELVIEW);
    glClearColor(0.0, 0.0, 0.0, 0.0);

    GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
    GLfloat light_ambient[] = {0.0, 0.0, 0.0, 1.0};
    GLfloat light_diffuse[] = {1.0, 1.0, 1.0, 1.0};
    GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
    GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
    GLfloat mat_specular[] = {1.0, 1.0, 1.0, 1.0};
    GLfloat mat_shininess[] = {10.0};
    glShadeModel(GL_SMOOTH);
    glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
    glLightfv(GL_LIGHT0, GL_POSITION, light_position);
    glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
    glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
    glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_COLOR_MATERIAL);
    glEnable(GL_DEPTH_TEST);

    glutPassiveMotionFunc(mousePassiveMotionCallback);
    glutMouseFunc(mymouse);
    glutDisplayFunc(Display);
    glutReshapeFunc(reshape);
    glutKeyboardFunc(KeyPressed);
    glutIdleFunc(idle);
    terminalPrint();
    glutMainLoop();

    // Cleanup resources
    movieOff();
    freeBodies();

    return 0;
}





