#include "hip/hip_runtime.h"
// nvcc nBodyArtB.cu -o nBodyArt -lglut -lm -lGLU -lGL																																							
//To stop hit "control c" in the window you launched it from.
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <signal.h>
#include <cmath>
#include <SOIL/SOIL.h>

using namespace std;

FILE* ffmpeg;


// defines for terminal stuff.
#define BOLD_ON  "\e[1m"
#define BOLD_OFF   "\e[m"
#define INITIAL_CAPACITY 100

FILE* MovieFile;

// Globals
int NumberOfInitBodies;
float TotalRunTime;
float Dt;
float G;
float H;
float Epsilon;
float MassOfBody;
float DiameterOfBody;
float VelocityMax;
float Drag;
int DrawRate;
int PrintRate;

// Other Globals
int Pause;
//float *BodyPositionX, *BodyPositionY, *BodyPositionZ;
//float *BodyVelocityX, *BodyVelocityY, *BodyVelocityZ;
//float *BodyForceX, *BodyForceY, *BodyForceZ;
//float *BodyColorX, *BodyColorY, *BodyColorZ;
int DrawTimer, PrintTimer;
float RunTime;
int* Buffer;
int MovieOn;
int MovieFlag;
int Trace;
float MouseX, MouseY, MouseZ;
float newBodyRadius = 0.1;
int DrawLayer = 0;
GLuint backgroundTexture;
float circleCenterX = 0.0f;
float circleCenterY = 0.0f;
float currentOscillationAmplitude = 0.0f;
float currentOscillationAngle = 0.0f;

float initialMouseX, initialMouseY;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize; 
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;



typedef struct
{
	int id;
	bool isSolid;
	float4 color;
	int movement; //preconfigured movement pattern
	float4 pos;
	float4 vel;
	float4 force;
	float radius;
    float initialX; //store the initial x position for sinusoidal movement
    float initialY; //store the initial y position for sinusoidal movement

    float4 circle; //(x,y) of the center of the circle for circular movement, z is initial angle, w is orbital radius
    float oscillationAmplitude; //amplitude of the oscillation for sinusoidal movement  
    float oscillationAngle; // Angle of the oscillation in radians

} Body;

// Prototyping functions
void setSimulationParameters();
void allocateMemory();
void setInitialConditions();
void drawPicture();
void nBody();
void errorCheck(const char*);
void terminalPrint();
void setup();
void movieOn();
void movieOff();
void screenShot();
float4 centerOfMass();
float4 linearVelocity();
void zeroOutSystem();
void addBody(Body newBody);

//Toggles
int NewBodyToggle = 0; // 0 if not currently adding a new body, 1 if currently adding a new body.
bool isOrthogonal = true;
int PreviousRunToggle = 1; // do you want to run a previous simulation or start a new one?
string PreviousRunFile = "test"; // The file name of the previous simulation you want to run.
int ColorToggle = 0; //15 possible values
int HotkeyPrint = 0; // 0 if not currently printing hotkeys, 1 if currently printing hotkeys.
int NewBodyMovement = 0; // 0 if random movement, 1 if circular movement
bool NewBodySolid = true; // 0 if not solid, 1 if solid
bool IsDragging = false;
bool GridOn = true;
bool EraseMode = false;
bool BackgroundToggle = true;
bool selectCircleCenter = false;


typedef struct //stores colors for Starry night
{
    float4 paris_m;
    float4 manz;
    float4 outer_space;
    float4 curious_blue;
    float4 tahuna_sands;
    float4 livid_brown;
    float4 neptune;
    float4 lochmara;
    float4 regal_blue;
    float4 vis_vis;
    float4 light_curious_blue;
    float4 ironside_grey;
    float4 yellow;
    float4 deco;
    float4 astronaut_blue;
    float4 bright_orange;
    //float4 fiery_red;
} Colors;

Colors colors = { // assigns values corresponding to the colors in the struct
    {49.0/255.0, 39.0/255.0, 96.0/255.0, 1.0},
    {228.0/255.0, 219.0/255.0, 85.0/255.0, 1.0},
    {65.0/255.0, 74.0/255.0, 76.0/255.0, 1.0},
    {21.18/255.0, 44.31/255.0, 77.65/255.0, 1.0},
    {93.0/255.0, 94.0/255.0, 78.0/255.0, 1.0},
    {49.0/255.0, 42.0/255.0, 41.0/255.0, 1.0},
    {49.0/255.0, 72.0/255.0, 73.0/255.0, 1.0},
    {50.0/255.0, 100.0/255.0, 150.0/255.0, 1.0},
    {14.0/255.0, 54.0/255.0, 87.0/255.0, 1.0},
    {249.0/255.0, 228.0/255.0, 150.0/255.0, 1.0},
    {15.0/255.0, 59.0/255.0, 82.0/255.0, 1.0},
    {40.0/255.0, 40.0/255.0, 38.0/255.0, 1.0},
    {244.0/255.0, 179.0/255.0, 5.0/255.0, 1.0},
    {198.0/255.0, 202.0/255.0, 116.0/255.0, 1.0},
    {42.0/255.0, 75.0/255.0, 124.0/255.0, 1.0},
    {240.0/255.0, 98.0/255.0, 16.0/255.0, 1.0},
};

float4 getColor(const char* colorName) { //to assign colors to the new body, call this function with the color name
    if (strcmp(colorName, "paris_m") == 0) return colors.paris_m;
    if (strcmp(colorName, "manz") == 0) return colors.manz;
    if (strcmp(colorName, "outer_space") == 0) return colors.outer_space;
    if (strcmp(colorName, "curious_blue") == 0) return colors.curious_blue;
    if (strcmp(colorName, "tahuna_sands") == 0) return colors.tahuna_sands;
    if (strcmp(colorName, "livid_brown") == 0) return colors.livid_brown;
    if (strcmp(colorName, "neptune") == 0) return colors.neptune;
    if (strcmp(colorName, "lochmara") == 0) return colors.lochmara;
    if (strcmp(colorName, "regal_blue") == 0) return colors.regal_blue;
    if (strcmp(colorName, "vis_vis") == 0) return colors.vis_vis;
    if (strcmp(colorName, "light_curious_blue") == 0) return colors.light_curious_blue;
    if (strcmp(colorName, "ironside_grey") == 0) return colors.ironside_grey;
    if (strcmp(colorName, "yellow") == 0) return colors.yellow;
    if (strcmp(colorName, "deco") == 0) return colors.deco;
    if (strcmp(colorName, "astronaut_blue") == 0) return colors.astronaut_blue;
    if (strcmp(colorName, "bright_orange") == 0) return colors.bright_orange;
    return (float4){0.0, 0.0, 0.0, 1.0}; // Default value
}

Body* bodies = NULL;
int numBodies = NumberOfInitBodies;
int capacity = INITIAL_CAPACITY; // Initial capacity of the bodies array


void readBodiesFromFile(const char* filename)
{
    // Append the file directory to the file
    string fileDir = "../PreviousRuns/";
    fileDir.append(filename); // Now fileDir = "/PreviousRuns/filename"

    FILE* file = fopen(fileDir.c_str(), "r");
    if (file == NULL)
    {
        fprintf(stderr, "Error: Could not open file %s for reading\n", filename);
        return;
    }

    // Read the number of bodies from the top of the file
    int numBodiesFromFile;
    if (fscanf(file, "Number of bodies: %d\n", &numBodiesFromFile) != 1)
    {
        fprintf(stderr, "Error: Could not read the number of bodies from the file\n");
        fclose(file);
        return;
    }

    // Allocate memory based on the number of bodies
    capacity = numBodiesFromFile;
    bodies = (Body*)malloc(capacity * sizeof(Body));
    if (bodies == NULL)
    {
        fprintf(stderr, "Memory allocation failed\n");
        fclose(file);
        exit(1);
    }
    printf("Initial memory allocated with capacity: %d\n", capacity);

    // Skip the header line
    char header[256];
    fgets(header, sizeof(header), file);

    // Read body information
    for (int i = 0; i < numBodiesFromFile; i++)
    {
        Body newBody;
        int isSolid;
        float color_x, color_y, color_z, color_w;
        float pos_x, pos_y, pos_z;
        float vel_x, vel_y, vel_z;
        float force_x, force_y, force_z;
        float initialX = 0.0f, initialY = 0.0f, oscillationAmplitude = 0.0f, oscillationAngle = 0.0f;
        int result = fscanf(file, "%d, %d, (%f, %f, %f, %f), %d, (%f, %f, %f), (%f, %f, %f), (%f, %f, %f), %f, %f, %f, %f, %f\n",
                            &newBody.id,
                            &isSolid,
                            &color_x, &color_y, &color_z, &color_w,
                            &newBody.movement,
                            &pos_x, &pos_y, &pos_z,
                            &vel_x, &vel_y, &vel_z,
                            &force_x, &force_y, &force_z,
                            &newBody.radius,
                            &initialX, &initialY, &oscillationAmplitude, &oscillationAngle);

        if (result == 17 || result == 21) // Old format or new format
        {
            newBody.isSolid = (bool)isSolid;
            newBody.color = make_float4(color_x, color_y, color_z, color_w);
            newBody.pos = make_float4(pos_x, pos_y, pos_z, 1.0f);
            newBody.vel = make_float4(vel_x, vel_y, vel_z, 0.0f);
            newBody.force = make_float4(force_x, force_y, force_z, 0.0f);

            if (result == 21) // New format
            {
                newBody.initialX = initialX;
                newBody.initialY = initialY;
                newBody.oscillationAmplitude = oscillationAmplitude;
                newBody.oscillationAngle = oscillationAngle;
            }
            else // Old format
            {
                newBody.initialX = 0.0f;
                newBody.initialY = 0.0f;
                newBody.oscillationAmplitude = 0.0f;
                newBody.oscillationAngle = 0.0f;
            }

            addBody(newBody);
            printf("Read body %d: id=%d, isSolid=%d, color=(%f, %f, %f, %f), movement=%d, pos=(%f, %f, %f), vel=(%f, %f, %f), force=(%f, %f, %f), radius=%f, initialX=%f, initialY=%f, oscillationAmplitude=%f, oscillationAngle=%f\n",
                   i, newBody.id, newBody.isSolid, newBody.color.x, newBody.color.y, newBody.color.z, newBody.color.w,
                   newBody.movement, newBody.pos.x, newBody.pos.y, newBody.pos.z,
                   newBody.vel.x, newBody.vel.y, newBody.vel.z,
                   newBody.force.x, newBody.force.y, newBody.force.z,
                   newBody.radius, newBody.initialX, newBody.initialY, newBody.oscillationAmplitude, newBody.oscillationAngle);
        }
        else
        {
            fprintf(stderr, "Error: fscanf read %d values instead of 17 or 21\n", result);
            break;
        }
    }

    fclose(file);
    printf("Body information read from %s\n", filename);

    // Update numBodies
    numBodies = numBodiesFromFile;
}

void writeBodiesToFile(const char* filename)
{
    string fileDir = "../PreviousRuns/";
    fileDir.append(filename); // Now fileDir = "/PreviousRuns/filename"

    FILE* file = fopen(fileDir.c_str(), "w");
    if (file == NULL)
    {
        fprintf(stderr, "Error: Could not open file %s for writing\n", filename);
        return;
    }

    // Write the number of bodies at the top of the file
    fprintf(file, "Number of bodies: %d\n", numBodies);

    // Write the header line
    fprintf(file, "ID, IsSolid, Color (R, G, B, A), Movement, Position (X, Y, Z), Velocity (X, Y, Z), Force (X, Y, Z), Radius, InitialX, InitialY, OscillationAmplitude, OscillationAngle\n");

    for (int i = 0; i < numBodies; i++)
    {
        fprintf(file, "%d, %d, (%f, %f, %f, %f), %d, (%f, %f, %f), (%f, %f, %f), (%f, %f, %f), %f, %f, %f, %f, %f\n",
                bodies[i].id,
                bodies[i].isSolid,
                bodies[i].color.x, bodies[i].color.y, bodies[i].color.z, bodies[i].color.w,
                bodies[i].movement,
                bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z,
                bodies[i].vel.x, bodies[i].vel.y, bodies[i].vel.z,
                bodies[i].force.x, bodies[i].force.y, bodies[i].force.z,
                bodies[i].radius,
                bodies[i].initialX, bodies[i].initialY, bodies[i].oscillationAmplitude, bodies[i].oscillationAngle);
    }

    fclose(file);
    printf("Body information written to %s\n", filename);
}

void addBody(Body newBody) 
{
    // Reallocate memory to accommodate the new body
	
    if (numBodies >= capacity) //if the new body will exceed the current capacity
	{
        capacity *= 2; //double the capacity
        Body* temp = (Body*)realloc(bodies, capacity * sizeof(Body)); //reallocate memory to accommodate the new body
        if (temp == NULL)  //if memory allocation fails
		{
            fprintf(stderr, "Memory allocation failed\n");
            exit(1);
        }
        bodies = temp;//assign the new memory to the bodies array, so long as memory allocation was successful
		//printf("Reallocated memory to capacity: %d\n", capacity);
    }


	//
	if(newBody.movement == 0) //random movement
	{
		newBody.vel.x = ((float)rand()/(float)RAND_MAX)*2.0f - 1.0f;
		newBody.vel.y = ((float)rand()/(float)RAND_MAX)*2.0f - 1.0f;
		newBody.vel.z = 0.0;
	}
	if (newBody.movement == 1) //still
	{
        newBody.vel.x = 0.0f;
        newBody.vel.y = 0.0f;
        newBody.vel.z = 0.0f;
	}
    if (newBody.movement == 2) //sinusoidal
    {
        newBody.vel.x = 0.2f;
        newBody.vel.y = 0.0f;
        newBody.vel.z = 0.0f;
        newBody.initialY = newBody.pos.y; // Store the initial y position
        //the rest needs to be done in nBody since it needs to be updated every frame
    }
    if (newBody.movement == 3) //circular
    {
        newBody.circle.x = circleCenterX; // Store the center x position
        newBody.circle.y = circleCenterY; // Store the center y position
        newBody.circle.z = atan2(newBody.pos.y - circleCenterY, newBody.pos.x - circleCenterX); // Calculate the initial angle
        newBody.circle.w = sqrt(pow(newBody.pos.x - circleCenterX, 2) + pow(newBody.pos.y - circleCenterY, 2)); // Calculate the radius
    }
    if (newBody.movement == 4) //oscillating
    {
        
        newBody.initialX = newBody.pos.x;
        newBody.initialY = newBody.pos.y;
        newBody.oscillationAmplitude = currentOscillationAmplitude; // Example amplitude
        newBody.oscillationAngle = currentOscillationAngle; // Example angle (0 radians for horizontal oscillation)
    }


    /// Add the new body to the array
	bodies[numBodies] = newBody;

    // Increment the number of bodies
    numBodies++;

	//for debugging
	//printf("Body %d added at (%f, %f, %f) with velocity (%f, %f, %f)\n", newBody.id, newBody.pos.x, newBody.pos.y, newBody.pos.z, newBody.vel.x, newBody.vel.y, newBody.vel.z);
}

void screenToWorld(int x, int y, float* worldX, float* worldY)
{
    float windowAspect = (float)XWindowSize / (float)YWindowSize;
    *worldX =  (2.0f * x / XWindowSize - 1.0f) * windowAspect * 3.0f + 1.1f;
    *worldY = (-2.0f * y / YWindowSize + 1.0f) * 1.5f - 0.5f;
    printf("Converted screen (%d, %d) to world (%f, %f)\n", x, y, *worldX, *worldY); // Debugging statement
}

void addBodyAtPosition(float x, float y)
{

    Body newBody;

    if (ColorToggle == 1)
    {
        newBody.color = getColor("paris_m");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 2)
    {
        newBody.color = getColor("manz");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 3)
    {
        newBody.color = getColor("outer_space");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 4)
    {
        newBody.color = getColor("curious_blue");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 5)
    {
        newBody.color = getColor("tahuna_sands");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 6)
    {
        newBody.color = getColor("livid_brown");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 7)
    {
        newBody.color = getColor("neptune");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 8)
    {
        newBody.color = getColor("lochmara");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 9)
    {
        newBody.color = getColor("regal_blue");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 10)
    {
        newBody.color = getColor("vis_vis");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 11)
    {
        newBody.color = getColor("light_curious_blue");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 12)
    {
        newBody.color = getColor("ironside_grey");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 13)
    {
        newBody.color = getColor("yellow");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 14)
    {
        newBody.color = getColor("deco");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 15)
    {
        newBody.color = getColor("astronaut_blue");
        HotkeyPrint = 0;
    }
    else if (ColorToggle == 16)
    {
        newBody.color = getColor("bright_orange");
        HotkeyPrint = 0;
    }
    else
    {
        newBody.color = {1.0f, 1.0f, 1.0f, 1.0f}; // default
    }

    newBody.id = numBodies;
    newBody.pos = make_float4(x, y, 0.0 + DrawLayer/100.0, 1.0f);
    newBody.vel = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    newBody.force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    newBody.radius = newBodyRadius *DiameterOfBody/2.0f;
    newBody.isSolid = NewBodySolid;
    newBody.movement = NewBodyMovement;

    addBody(newBody);
    printf("Added body at (%f, %f)\n", x, y); // Debugging statement
}

void removeBodyAtPosition(float x, float y)
{
    for (int i = 0; i < numBodies; ++i)
    {
        float dx = bodies[i].pos.x - x;
        float dy = bodies[i].pos.y - y;
        float distance = sqrt(dx * dx + dy * dy);

        if (distance < bodies[i].radius)
        {
            // Remove the body by shifting the remaining bodies
            for (int j = i; j < numBodies - 1; ++j)
            {
                bodies[j] = bodies[j + 1];
            }
            --numBodies;
            printf("Removed body at (%f, %f)\n", x, y); // Debugging statement
            return;
        }
    }
}


void freeBodies() 
{
    free(bodies);
}

void drawGrid(float spacing, int numLines)
{
    glColor3f(0.8f, 0.8f, 0.8f); // Set grid color (light gray)
    glBegin(GL_LINES);

    // Draw vertical lines
    for (int i = -numLines; i <= numLines; ++i)
    {
        float x = i * spacing;
        glVertex3f(x, -numLines * spacing, 0.0f);
        glVertex3f(x, numLines * spacing, 0.0f);
    }

    // Draw horizontal lines
    for (int i = -numLines; i <= numLines; ++i)
    {
        float y = i * spacing;
        glVertex3f(-numLines * spacing, y, 0.0f);
        glVertex3f(numLines * spacing, y, 0.0f);
    }

    glEnd();
}

void setup()
{
	allocateMemory();
    if (PreviousRunToggle == 1)
    {
        // Read the previous simulation parameters from the specified file
        readBodiesFromFile(PreviousRunFile.c_str());
        setSimulationParameters();
        //zeroOutSystem();
    }
    else
    {
        // Set up a new simulation
        setSimulationParameters();
        allocateMemory();
        setInitialConditions();
        zeroOutSystem();
    }

    DrawTimer = 0;
    PrintRate = 0;
    RunTime = 0.0;
    Trace = 0;
    Pause = 1;
    MovieOn = 0;
    terminalPrint();
}

void Display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	drawPicture();
	glutSwapBuffers();
}

void idle()
{
    if (NewBodyToggle == 1)
    {
        drawPicture();
    }
    else
    {
        nBody();
    }
}

void reshape(int w, int h)
{
    // Prevent division by zero
    if (h == 0) h = 1;

    // Calculate the aspect ratio of the window
    float aspectRatio = (float)w / (float)h; //currently 3000/1500 = 2

    // Set the viewport to cover the new window
    glViewport(0, 0, (GLsizei)w, (GLsizei)h);

    // Set the projection matrix
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    // Adjust the projection matrix to maintain the aspect ratio of the bodies
    if (isOrthogonal) 
	{
        if (aspectRatio >= 1.0f) 
		{
            // Window is wider than it is tall
            glOrtho(-1.0 * aspectRatio, 1.0 * aspectRatio, -1.0, 1.0, Near, Far);
        } 
		else 
		{
            // Window is taller than it is wide
            glOrtho(-1.0, 1.0, -1.0 / aspectRatio, 1.0 / aspectRatio, Near, Far);
        }
    } 
	else 
	{
        if (aspectRatio >= 1.0f) 
		{
            // Window is wider than it is tall
            glFrustum(-0.2 * aspectRatio, 0.2 * aspectRatio, -0.2, 0.2, Near, Far);
        } else 
		{
            // Window is taller than it is wide
            glFrustum(-0.2, 0.2, -0.2 / aspectRatio, 0.2 / aspectRatio, Near, Far);
        }
    }

    // Switch back to the modelview matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
}

void KeyPressed(unsigned char key, int x, int y)
{	
	if(key == 'q')
	{
		// Check if ffmpeg is not NULL before closing
        if (ffmpeg != NULL) 
		{
            pclose(ffmpeg);
            ffmpeg = NULL; // Optionally set to NULL after closing
        } 
		else 
		{
            fprintf(stderr, "Warning: Attempted to close a NULL file pointer\n");
        }
        glutDestroyWindow(Window);
        printf("\nw Good Bye\n");
        exit(0);
	}
	// if(key == 'v') //not much need for this anymore
   	// {
    //     // Toggle the view mode
    //     isOrthogonal = !isOrthogonal;

    //     // Call reshape to update the projection matrix
    //     reshape(glutGet(GLUT_WINDOW_WIDTH), glutGet(GLUT_WINDOW_HEIGHT));

    //     // Redraw the scene
    //     glutPostRedisplay();
   	//  }
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 't') // Turns tracers on and off
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 'M')  // Movie on/off
	{
		if(MovieFlag == 0) 
		{
			MovieFlag = 1;
			movieOn();
		}
		else 
		{
			MovieFlag = 0;
			movieOff();
		}
		terminalPrint();
	}
	
	if(key == 'S')  // Screenshot
	{	
		screenShot();
		terminalPrint();
	}
	if (key == 'n') // Add a new body
	{
		if(NewBodyToggle == 0) NewBodyToggle = 1;
		else NewBodyToggle = 0;
		terminalPrint();
	}
	if(key == ']')  
	{
		newBodyRadius += 0.01;
		terminalPrint();
		//printf("\n Your selection area = %f times the radius of atrium. \n", HitMultiplier);
	}
	if(key == '[')
	{
		newBodyRadius -= 0.01;
		if(newBodyRadius < 0.0) newBodyRadius = 0.0;
		terminalPrint();
		//printf("\n Your selection area = %f times the radius of atrium. \n", HitMultiplier);
	}
	if(key == 's')
	{
        printf("Enter the file name to save this run to: ");
        char filename[256];
        scanf("%s", filename);
        writeBodiesToFile(filename);
	}
    if(key == 'e')
    {
        if(EraseMode)
        {
            EraseMode = false;
            terminalPrint();
        }
        else
        {
            EraseMode = true;
            terminalPrint();
        }
    }
    if(key == 'g')
    {
        if(GridOn)
        {
            GridOn = false;
            drawPicture();
            terminalPrint();
        }
        else
        {
            GridOn = true;
            drawPicture();
            terminalPrint();
        }
    }

    if(key == 'b')
    {
        if(BackgroundToggle)
        {
            BackgroundToggle = false;
            drawPicture();
            terminalPrint();
        }
        else
        {
            BackgroundToggle = true;
            drawPicture();
            terminalPrint();
        }
    }

    if(NewBodyToggle == 1)
    {
        if (key == 'l') // cycle through colors, forward
        {
            if (ColorToggle < 16)
            {
                ColorToggle++;
            }
            else
            {
                ColorToggle = 1;
            }
            terminalPrint();
        }
        if (key == 'k') // cycle through colors, backward
        {
            if (ColorToggle > 1)
            {
                ColorToggle--;
            }
            else
            {
                ColorToggle = 15;
            }
            terminalPrint();
        }
        //set movement pattern
        //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!CHANGES NEED TO BE MADE LATER!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        if (key == 'm')
        {
            printf("Enter the movement pattern for the new body: ");
            printf("0 for random movement, 1 for still, 2 for sinusoidal, 3 for circular, 4 for oscillating\n");
            scanf("%d", &NewBodyMovement);
            if (NewBodyMovement < 0 || NewBodyMovement > 4)
            {
                printf("Invalid movement pattern.\n");
                NewBodyMovement = 0;
            }
            if(NewBodyMovement == 3)
            {
                //get the center of the circle from the user using the mouse
                selectCircleCenter = true;
                currentOscillationAmplitude = 0.0f;
            }
            
            terminalPrint();
        }

        if(key == 'i')//is the new body solid?
        {
            if(NewBodySolid == true)
            {
                NewBodySolid = false;
            }
            else
            {
                NewBodySolid = true;
            }
            terminalPrint();
        }

        //add DrawLayer so you can decide what appears on top of what
        if(key == 'u')
        {
            DrawLayer++;
            drawPicture();
            terminalPrint();
        }
        if(key == 'y')
        {
            DrawLayer--;
            drawPicture();
            terminalPrint();
        }

       if(NewBodyMovement == 4)
       {
            if (key == 'r') // Rotate oscillation angle left
            {
                currentOscillationAngle -= 0.1f; // Adjust the angle increment as needed
                if (currentOscillationAngle < 0.0f)
                {
                    currentOscillationAngle += 2.0f * M_PI;
                }
                drawPicture();
                terminalPrint();
            }
            if (key == 'R') // Rotate oscillation angle right
            {
                currentOscillationAngle += 0.1f; // Adjust the angle increment as needed
                if (currentOscillationAngle >= 2.0f * M_PI)
                {
                    currentOscillationAngle -= 2.0f * M_PI;
                }
                drawPicture();
                terminalPrint();
            }

            if (key == '+') // Increase oscillation amplitude
            {
                currentOscillationAmplitude += 0.01f; // Adjust the amplitude increment as needed
                drawPicture();
                terminalPrint();
            }
            if (key == '-') // Decrease oscillation amplitude
            {
                currentOscillationAmplitude -= 0.01f; // Adjust the amplitude increment as needed
                if (currentOscillationAmplitude < 0.0f)
                {
                    currentOscillationAmplitude = 0.0f;
                }
                drawPicture();
                terminalPrint();
            }
       }
    }
}

void mousePassiveMotionCallback(int x, int y) 
{


    float windowAspect = (float)XWindowSize / (float)YWindowSize;
    MouseX = (5.76 * x / XWindowSize) - 1.84f; // Map x to (-1.84, 1.84)
    MouseY = -(2.9f * y / YWindowSize) + 1.0f;   // Map y to (-1, 1)
    MouseZ = 0.0f;
    if (IsDragging)
    {
        if(EraseMode)
        {
            removeBodyAtPosition(MouseX, MouseY);
        }
        else
        {
            addBodyAtPosition(MouseX, MouseY);
        }
    }


    // Redraw the scene
    //glutPostRedisplay();
    // Print the converted coordinates for debugging
    //printf("MouseX: %f, MouseY: %f\n", MouseX, MouseY);
}

// This is called when you push a mouse button.
void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{	
		if(button == GLUT_LEFT_BUTTON)
		{	
			if(NewBodyToggle == 1)
			{
                if(EraseMode)
                {
                    removeBodyAtPosition(MouseX, MouseY);
                }
                else if(selectCircleCenter)
                {
                    // Convert screen coordinates to world coordinates
                    screenToWorld(x, y, &circleCenterX, &circleCenterY);
                    printf("Circle center selected at (%f, %f)\n", circleCenterX, circleCenterY);
                    selectCircleCenter = false; // Reset the flag
                }
                else
                {
                    //generate random numbers for all the properties of the new body
                    
                    int index = numBodies; // Define and initialize index

                    // Convert window coordinates to OpenGL coordinates
                    float windowAspect = (float)XWindowSize / (float)YWindowSize;
                    MouseX = (5.76 * x / XWindowSize) - 1.84f; // Map x to (-1.84, 1.84)
                    MouseY = -(2.9f * y / YWindowSize) + 1.0f;   // Map y to (-1, 1)
                    MouseZ = 0.0f;
                    MouseZ = 0.0f;

                    // Print the converted coordinates for debugging
                    printf("MouseX: %f, MouseY: %f, MouseZ: %f\n", MouseX, MouseY, MouseZ);

                    Body newBody; //create a new body with the body struct

                    // Set the color of the new body based on the ColorToggle

                    if(ColorToggle == 1)
                    {
                        newBody.color = getColor("paris_m");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 2)
                    {
                        newBody.color = getColor("manz");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 3)
                    {
                        newBody.color = getColor("outer_space");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 4)
                    {
                        newBody.color = getColor("curious_blue");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 5)
                    {
                        newBody.color = getColor("tahuna_sands");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 6)
                    {
                        newBody.color = getColor("livid_brown");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 7)
                    {
                        newBody.color = getColor("neptune");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 8)
                    {
                        newBody.color = getColor("lochmara");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 9)
                    {
                        newBody.color = getColor("regal_blue");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 10)
                    {
                        newBody.color = getColor("vis_vis");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 11)
                    {
                        newBody.color = getColor("light_curious_blue");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 12)
                    {
                        newBody.color = getColor("ironside_grey");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 13)
                    {
                        newBody.color = getColor("yellow");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 14)
                    {
                        newBody.color = getColor("deco");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 15)
                    {
                        newBody.color = getColor("astronaut_blue");
                        HotkeyPrint = 0;
                    }
                      else if(ColorToggle == 16)
                    {
                        newBody.color = getColor("bright_orange");
                        HotkeyPrint = 0;
                    }
                    else
                    {
                        newBody.color =  {1.0f, 1.0f, 1.0f, 1.0f}; //default
                    }

                    //assign all the properties of the new body
                    newBody.id = index;
                    newBody.isSolid = true;
                    newBody.movement = NewBodyMovement;
                    newBody.pos = {MouseX, MouseY, MouseZ + DrawLayer/100.0f, 1.0f}; // Directly assign values to float4
                    newBody.force = {0.0f, 0.0f, 0.0f, 0.0f}; // Directly assign values to float4
                    newBody.radius = newBodyRadius * DiameterOfBody/2.0f;

                    addBody(newBody);
                }   
            }
		}
		else if(button == GLUT_RIGHT_BUTTON) // Right Mouse button down
		{
            if (state == GLUT_DOWN)
            {
                //make it a toggle
                if(IsDragging == false)
                {
                    IsDragging = true;
                    float windowAspect = (float)XWindowSize / (float)YWindowSize;
                    MouseX = (5.76 * x / XWindowSize) - 1.84f; // Map x to (-1.84, 1.84)
                    MouseY = -(2.9f * y / YWindowSize) + 1.0f;   // Map y to (-1, 1)
                    MouseZ = 0.0f;
                }
                else
                {
                    IsDragging = false;
                }
            }
            else if (state == GLUT_UP)
            {
                IsDragging = false;
                printf("Mouse up at (%f, %f)\n", MouseX, MouseY); // Debugging statement
            }
		}
		else if(button == GLUT_MIDDLE_BUTTON)
		{
			// Do stuff in here if you choose to when the middle mouse button is pressed.
		}
	}
	
	// If no mouse button is down (state 0, they don't have a nice word like GLUT_NOT_DOWN) 
	// but you move the mouse wheel this is called.
	if(state == 0)
	{
		// When you turn the mouse whell forward this is called.
		if(button == 3)
		{
			EyeZ -=0.1;
			
		}
		
		// When you turn the mouse whell backward this is called.
		else if(button == 4)
		{
			EyeZ += 0.1;
			
		}
	}
	glLoadIdentity();
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	//glutPostRedisplay();
}

void loadBackgroundImage(const char* filename)
{
    backgroundTexture = SOIL_load_OGL_texture(
        filename,
        SOIL_LOAD_AUTO,
        SOIL_CREATE_NEW_ID,
        SOIL_FLAG_INVERT_Y
    );

    if (backgroundTexture == 0)
    {
        printf("SOIL loading error: '%s'\n", SOIL_last_result());
    }
}

void renderBackground()
{
    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, backgroundTexture);

    // Save the current color state
    GLboolean colorMask[4];
    glGetBooleanv(GL_COLOR_WRITEMASK, colorMask);
    GLfloat currentColor[4];
    glGetFloatv(GL_CURRENT_COLOR, currentColor);

    // Reset color to white
    glColor3f(1.0f, 1.0f, 1.0f);

    // Calculate aspect ratio
    float windowAspect = (float)XWindowSize / (float)YWindowSize;

    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex3f(-windowAspect, -1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex3f(windowAspect, -1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex3f(windowAspect, 1.0f, -1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex3f(-windowAspect, 1.0f, -1.0f);
    glEnd();

    glDisable(GL_TEXTURE_2D);

    // Restore the previous color state
    glColor4fv(currentColor);
    glColorMask(colorMask[0], colorMask[1], colorMask[2], colorMask[3]);
}

string getTimeStamp()
{
	// Want to get a time stamp string representing current date/time, so we have a
	// unique name for each video/screenshot taken.
	time_t t = time(0); 
	struct tm * now = localtime( & t );
	int month = now->tm_mon + 1, day = now->tm_mday, year = now->tm_year, 
				curTimeHour = now->tm_hour, curTimeMin = now->tm_min, curTimeSec = now->tm_sec;
	stringstream smonth, sday, syear, stimeHour, stimeMin, stimeSec;
	smonth << month;
	sday << day;
	syear << (year + 1900); // The computer starts counting from the year 1900, so 1900 is year 0. So we fix that.
	stimeHour << curTimeHour;
	stimeMin << curTimeMin;
	stimeSec << curTimeSec;
	string timeStamp;
	if (curTimeMin <= 9)	
		timeStamp = smonth.str() + "-" + sday.str() + "-" + syear.str() + '_' + stimeHour.str() + ".0" + stimeMin.str() + 
					"." + stimeSec.str();
	else			
		timeStamp = smonth.str() + "-" + sday.str() + '-' + syear.str() + "_" + stimeHour.str() + "." + stimeMin.str() +
					"." + stimeSec.str();
	return timeStamp;
}

// Signal handler for SIGPIPE
void handle_sigpipe(int sig)
{
    fprintf(stderr, "Caught SIGPIPE signal: %d\n", sig);
}

void movieOn()
{
    // Register the SIGPIPE signal handler
    signal(SIGPIPE, handle_sigpipe);

    string ts = getTimeStamp();
    ts.append(".mp4");

    // Convert the x and y window size to a string of format "XsizexYsize"
    stringstream ss;
    ss << XWindowSize << "x" << YWindowSize;
    string windowSize = ss.str();

    // Setting up the movie buffer with the dynamic window size
    string baseCommand = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s " + windowSize + " -i - "
                         "-c:v libx264rgb -threads 0 -preset fast -y -pix_fmt yuv420p -crf 0 -vf vflip 2>ffmpeg_error.log ";

    string z = baseCommand + ts;

    const char *ccx = z.c_str();
    MovieFile = popen(ccx, "w");

    // Check if popen was successful
    if (MovieFile == NULL) {
        fprintf(stderr, "Error: Failed to open movie file with popen\n");
        return;
    }

    // Allocate buffer
    Buffer = (int*)malloc(XWindowSize * YWindowSize * sizeof(int));

    // Check if malloc was successful
    if (Buffer == NULL) {
        fprintf(stderr, "Error: Failed to allocate memory for buffer\n");
        pclose(MovieFile);
        MovieFile = NULL;
        return;
    }

    MovieOn = 1;
    printf("Movie recording started successfully\n");
}

void movieOff()
{
	if(MovieOn == 1) 
	{
		pclose(MovieFile);
	}
	free(Buffer);
	MovieOn = 0;
}

void screenShot()
{	
	int pauseFlag;
	FILE* ScreenShotFile;
	int* buffer;

	//convert the x and y windowsize to a string of format "XsizexYsize"
    stringstream ss;
    ss << XWindowSize << "x" << YWindowSize;
    string windowSize = ss.str();

    // Construct the ffmpeg command with the dynamic window size
    string baseCommand = "ffmpeg -loglevel quiet -framerate 60 -f rawvideo -pix_fmt rgba -s " + windowSize + " -i - "
                         "-c:v libx264rgb -threads 0 -preset fast -y -crf 0 -vf vflip output1.mp4";
    const char* cmd = baseCommand.c_str();
	//const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
	//              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
	ScreenShotFile = popen(cmd, "w");
	buffer = (int*)malloc(XWindowSize*YWindowSize*sizeof(int));
	
	if(Pause == 0) 
	{
		Pause = 1;
		pauseFlag = 0;
	}
	else
	{
		pauseFlag = 1;
	}
	
	for(int i =0; i < 1; i++)
	{
		drawPicture();
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
		fwrite(buffer, sizeof(int)*XWindowSize*YWindowSize, 1, ScreenShotFile);
	}
	
	pclose(ScreenShotFile);
	free(buffer);

	string ts = getTimeStamp(); // Only storing in a separate variable for debugging purposes.
	string s = "ffmpeg -loglevel quiet -i output1.mp4 -qscale:v 1 -qmin 1 -qmax 1 " + ts + ".jpeg";
	// Convert back to a C-style string.
	const char *ccx = s.c_str();
	system(ccx);
	system("rm output1.mp4");
	printf("\nScreenshot Captured: \n");
	cout << "Saved as " << ts << ".jpeg" << endl;
	
	Pause = pauseFlag;
	//ffmpeg -i output1.mp4 output_%03d.jpeg
}

void setSimulationParameters()
{
    if(PreviousRunToggle  == 0) numBodies = 0; //start with no bodies, a blank canvas

	TotalRunTime = 10000.0;

	Dt = 0.002;

	// This is a lennard-Jones type force G*m1*m2/(r^2) - H*m1*m2/(r^4).
	// If you want a gravity type force just set G to your gravity and set H equal 0.
	G = 0.03;

	H = 0.00001;

	Epsilon = 0.01;

	MassOfBody = 1.0;

	DiameterOfBody = 0.2;

	VelocityMax = 10.0;

	Drag = 0.001;

	DrawRate = 8;
	
	PrintRate = 100;
}

void allocateMemory()
{
// Allocate initial memory for the bodies array
    bodies = (Body*)malloc(capacity * sizeof(Body));
    if (bodies == NULL) 
    {
        fprintf(stderr, "Initial memory allocation failed\n");
        exit(1);
    }
    printf("Initial memory allocated with capacity: %d\n", capacity);
}

void setInitialConditions()
{
    float dx, dy, dz, d, d2;
    int test;
	time_t t;
	
	srand((unsigned) time(&t));
	for(int i = 0; i < numBodies; i++)
	{
		bodies[i].id = i;
		test = 0;
		while(test == 0)
		{
			// Get random number between -1 at 1.
			bodies[i].pos.x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			bodies[i].pos.y= ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			bodies[i].pos.z= 0.0;  //((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			test = 1;
			
			for(int j = 0; j < i; j++)
			{
				dx = bodies[i].pos.x - bodies[j].pos.x;
				dy = bodies[i].pos.y - bodies[j].pos.y;
				dz = bodies[i].pos.z - bodies[j].pos.z;
				d2  = dx*dx + dy*dy + dz*dz;
				d = sqrt(d2);
				if(d < DiameterOfBody)
				{
					test = 0;
					break;
				}
			}
			
			if(test == 1)
			{
				bodies[i].vel.x = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				bodies[i].vel.y = 0.0; //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				bodies[i].vel.z = 0.0;  //VelocityMax*((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
				
				bodies[i].color.x = ((float)rand()/(float)RAND_MAX);
				bodies[i].color.y = ((float)rand()/(float)RAND_MAX);
				bodies[i].color.z = ((float)rand()/(float)RAND_MAX);
			}
		}
		//set the radius of the body
		bodies[i].radius =((float)rand()/(float)RAND_MAX)* DiameterOfBody/2.0;

		//initialize everything else to zero
		bodies[i].force.x = 0.0;
		bodies[i].force.y = 0.0;
		bodies[i].force.z = 0.0;
		bodies[i].movement = 0;
		bodies[i].isSolid = true;
		
	}
}

float4 centerOfMass()
{
	float totalMass;
	float4 centerOfMass;
	
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < numBodies; i++)
	{
    	centerOfMass.x += bodies[i].pos.x*MassOfBody;
		centerOfMass.y += bodies[i].pos.y*MassOfBody;
		centerOfMass.z += bodies[i].pos.z*MassOfBody;
		totalMass += MassOfBody;
	}
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	
	return(centerOfMass);
}

float4 linearVelocity()
{
	float totalMass;
	float4 linearVelocity;
	
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	totalMass = 0.0;
	
	for(int i = 0; i < numBodies; i++)
	{
    	linearVelocity.x += bodies[i].vel.x*MassOfBody;
		linearVelocity.y += bodies[i].vel.y*MassOfBody;
		linearVelocity.z += bodies[i].vel.z*MassOfBody;
		totalMass += MassOfBody;
	}
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	
	return(linearVelocity);
}

void zeroOutSystem()
{
	float4 pos, vel;
	pos = centerOfMass();
	vel = linearVelocity();
		
	for(int i = 0; i < numBodies; i++)
	{
		bodies[i].pos.x -= pos.x;
		bodies[i].pos.y -= pos.y;
		bodies[i].pos.z -= pos.z;
		
		bodies[i].vel.x -= vel.x;
		bodies[i].vel.y -= vel.y;
		bodies[i].vel.z -= vel.z;
	}
}

void drawPicture()
{
    if (Trace == 0)
    {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    }

    // Render the background image
    if (BackgroundToggle && backgroundTexture != 0)
    {
        renderBackground();
    }

    if (GridOn)
    {
        drawGrid(0.1f, 19); // Adjust spacing and number of lines as needed
    }

    if (NewBodyToggle == 1)
    {
        float4 mouseColor;

        if (ColorToggle == 1)
        {
            //color paris m
            mouseColor = getColor("paris_m");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 2)
        {
            //color manz
            mouseColor = getColor("manz");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 3)
        {
            //color outer space
            mouseColor = getColor("outer_space");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 4)
        {
            //color curious blue
            mouseColor = getColor("curious_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 5)
        {
            //color tahuna sands
            mouseColor = getColor("tahuna_sands");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 6)
        {
            //color livid brown
            mouseColor = getColor("livid_brown");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 7)
        {
            //color neptune
            mouseColor = getColor("neptune");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 8)
        {
            //color lochmara
            mouseColor = getColor("lochmara");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 9)
        {
            //color regal blue
            mouseColor = getColor("regal_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 10)
        {
            //color vis vis
            mouseColor = getColor("vis_vis");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 11)
        {
            //color light curious blue
            mouseColor = getColor("light_curious_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 12)
        {
            //color ironside grey
            mouseColor = getColor("ironside_grey");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 13)
        {
            //color yellow
            mouseColor = getColor("yellow");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 14)
        {
            //color deco
            mouseColor = getColor("deco");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 15)
        {
            //color astronaut blue
            mouseColor = getColor("astronaut_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 16)
        {
            //color astronaut blue
            mouseColor = getColor("bright_orange");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else
        {
            //color white
            glColor3d(1.0, 1.0, 1.0);
        }
        glPushMatrix();
        glTranslatef(MouseX, MouseY, MouseZ + DrawLayer / 100.0f);
        glutSolidSphere(newBodyRadius * DiameterOfBody / 2.0, 20, 20);
        glPopMatrix();

        if (NewBodyMovement == 4)
        {
            float dx = currentOscillationAmplitude * cos(currentOscillationAngle);
            float dy = currentOscillationAmplitude * sin(currentOscillationAngle);
            glColor3f(1.0f, 0.0f, 0.0f); // Red color for the line
            glBegin(GL_LINES);
            glVertex3f(MouseX, MouseY, MouseZ);
            glVertex3f(MouseX + dx, MouseY + dy, MouseZ);
            glEnd();
        }
    }

    for (int i = 0; i < numBodies; i++)
    {
        glColor3d(bodies[i].color.x, bodies[i].color.y, bodies[i].color.z);
        glPushMatrix();
        glTranslatef(bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z);
        glutSolidSphere(bodies[i].radius, 20, 20);
        glPopMatrix();
    }

    glutSwapBuffers();

    if (MovieOn == 1)
    {
        glReadPixels(0, 0, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
        fwrite(Buffer, sizeof(int) * XWindowSize * YWindowSize, 1, MovieFile);
    }

 
}

void getForces(Body* bodies, float mass, float G, float H, float Epsilon, float drag, float dt, int n)
{
	float dx, dy, dz, d2, d;
    float forceMag;
    float inOut;
	float kSphereReduction = 0.5;
	float dvx, dvy, dvz;
	float kSphere = 10000;

    // Initialize forces to zero
    for (int i = 0; i < n; i++)
    {
        bodies[i].force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }
    
    // Calculate forces
    for (int i = 0; i < n; i++)
    {
        if(bodies[i].movement != 1) //if the body is not still (movement 1 is still)
        {

        
            for (int j = i + 1; j < n; j++)
            {
                dx = bodies[j].pos.x - bodies[i].pos.x;
                dy = bodies[j].pos.y - bodies[i].pos.y;
                dz = bodies[j].pos.z - bodies[i].pos.z;
                d2 = dx * dx + dy * dy + dz * dz + Epsilon;
                d = sqrt(d2);
                if (d < 1e-6) 
                {
                    fprintf(stderr, "Warning: Small distance in force calculation, skipping\n");
                    continue;
                }
                //forceMag = (G * mass * mass) / d2 - (H * mass * mass) / (d2 * d2); // gravitational force
                forceMag = 0.0; //No force between bodies. Each body acts individually.

                float3 force = make_float3(forceMag * dx / d,
                                        forceMag * dy / d,
                                        forceMag * dz / d);

                if(bodies[i].isSolid ^ bodies[j].isSolid) //bitwise XOR. If one is solid and the other is not, and only then, do the following.
                {
                    float combinedDiamter = bodies[i].radius + bodies[j].radius;
                    if(d < combinedDiamter) //if the balls touch. i.e if the distance betweeen < both radii
                    {
                        
                        dvx = bodies[j].vel.x - bodies[i].vel.x;
                        dvy = bodies[j].vel.y - bodies[i].vel.y;
                        dvz = bodies[j].vel.z - bodies[i].vel.z;
                        inOut = dx*dvx + dy*dvy + dz*dvz;
                        if(inOut < 0.0) forceMag = kSphere*(combinedDiamter - d); // If inOut is negative the sphere are converging.
                        else forceMag = kSphereReduction*kSphere*(combinedDiamter - d); // If inOut is positive the sphere are diverging.
                        
                        // Doling out the force in the proper perfortions using unit vectors.
                        bodies[i].force.x -= forceMag*(dx/d);
                        bodies[i].force.y -= forceMag*(dy/d);
                        bodies[i].force.y -= forceMag*(dz/d);
                        // A force on me causes the opposite force on you. 
                        bodies[j].force.x += forceMag*(dx/d);
                        bodies[j].force.y += forceMag*(dy/d);
                        bodies[j].force.z += forceMag*(dz/d);
                    }
                }

                bodies[i].force.x += force.x;
                bodies[i].force.y += force.y;
                bodies[i].force.z += force.z;

                bodies[j].force.x -= force.x;
                bodies[j].force.y -= force.y;
                bodies[j].force.z -= force.z;
            }
        }
    }
	
}

void nBody()
{
    if (Pause != 1)
    {
        // Update positions and velocities
        for (int i = 0; i < numBodies; i++)
        {
            if (bodies[i].movement == 2) // sinusoidal
            {
                float frequency = 1.0f; // Adjust this value to change the period of the sine wave
                float amplitude = 0.2f; // Adjust this value to change the amplitude of the sine wave
    
                bodies[i].pos.x += bodies[i].vel.x * Dt;
                bodies[i].pos.y = bodies[i].initialY + amplitude * sin(frequency * bodies[i].pos.x);
            }
            else if (bodies[i].movement == 3) // circular
            {
                float angularVelocity = 2.0f; // Adjust this value to change the angular velocity of the circle

                float angle = bodies[i].circle.z + angularVelocity * RunTime;
                bodies[i].pos.x = bodies[i].circle.x + bodies[i].circle.w * cos(angle);
                bodies[i].pos.y = bodies[i].circle.y + bodies[i].circle.w * sin(angle);
            }
            else if (bodies[i].movement == 4) // Oscillation movement
            {
                float time = RunTime; // Use the elapsed time for smooth oscillation
                float frequency = 1.0f; // Adjust this value to change the frequency of the oscillation
                float amplitude = bodies[i].oscillationAmplitude; // Use the amplitude set for the body
                float angle = bodies[i].oscillationAngle; // Use the angle set for the body

                // Calculate the new position using a sine function
                bodies[i].pos.x = bodies[i].initialX + amplitude * cos(angle) * sin(frequency * time);
                bodies[i].pos.y = bodies[i].initialY + amplitude * sin(angle) * sin(frequency * time);

                // Debugging statements
                //printf("Body %d: pos=(%f, %f), initial=(%f, %f), amplitude=%f, frequency=%f, angle=%f, time=%f\n",
                //     bodies[i].id, bodies[i].pos.x, bodies[i].pos.y, bodies[i].initialX, bodies[i].initialY, amplitude, frequency, angle, time);
            }
            else
            {
                // Update position based on velocity for other movement types
                bodies[i].pos.x += bodies[i].vel.x * Dt;
                bodies[i].pos.y += bodies[i].vel.y * Dt;
                bodies[i].pos.z += bodies[i].vel.z * Dt;
            }
        }

        DrawTimer++;
        if (DrawTimer == DrawRate)
        {
            drawPicture();
            DrawTimer = 0;
        }

        PrintTimer++;
        if (PrintTimer == PrintRate)
        {
            // Print information if needed
            PrintTimer = 0;
        }

        RunTime += Dt;
        if (TotalRunTime < RunTime)
        {
            Pause = 1;
        }
    }
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	//system("clear");
	
    printf("\n");
	printf("\n S: Screenshot");
	
	printf("\n");
	printf("\n q: Terminates the simulation");

    printf("\n");
    printf("\n s: Save this run");

	printf("\n\n");
	printf("\033[0m");
	printf(" p: Pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf("\n");
	printf("\033[0m");
	printf(" t: Trace on/off toggle --> ");
	printf(" Trace is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;31mOn\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mOff\n" "\e[m");
	}
	//printf("\n");
	//printf("\033[0m");
	//printf(" v: Toggle view (Perspective/Orthogonal) --> ");
	//printf(" Current View: ");
	// if (isOrthogonal) 
	// {
	// 	printf("\e[1m" " \033[0;32mOrthogonal\n" "\e[m");
	// }
	// else 
	// {
	// 	printf("\e[1m" " \033[0;31mDefault\n" "\e[m");
	// }
	printf("\n M: Video On/Off toggle --> ");
	if (MovieFlag == 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "Video Recording Off\n" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Video Recording On\n" BOLD_OFF);
	}
    printf("\n");
    printf("\033[0m");
    printf(" g: Grid On/Off Toggle --> ");
    if (GridOn)
    {
        printf("\033[0;32m");
        printf(BOLD_ON "Grid On" BOLD_OFF);
    }
    else
    {
        printf("\033[0;31m");
        printf(BOLD_ON "Grid Off" BOLD_OFF);
    }

    printf("\n");
    printf("\033[0m");
    printf(" b: Background On/Off Toggle --> ");
    if (BackgroundToggle)
    {
        printf("\033[0;32m");
        printf(BOLD_ON "Background On" BOLD_OFF);
    }
    else
    {
        printf("\033[0;31m");
        printf(BOLD_ON "Background Off" BOLD_OFF);
    }



	printf("\n n: Simulaton Mode Add View/Add Body Toggle --> Mode:");
	if (NewBodyToggle== 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "View" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Add Body" BOLD_OFF);
	}
	//controls for body placement
    if(NewBodyToggle == 1)
    {
        printf("\n");
        printf("\033[0m");
        printf(" [/]: Change radius of new body backwards/forwards\n");

        printf("\n");
        printf("\033[0m");
        printf(" k/l: Change color of new body backwards/forwards\n");
        printf(" Current Color: ");
        if (ColorToggle == 1)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Paris M" BOLD_OFF);
        }
        else if (ColorToggle == 2)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Manz" BOLD_OFF);
        }
        else if (ColorToggle == 3)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Outer Space" BOLD_OFF);
        }
        else if (ColorToggle == 4)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Curious Blue" BOLD_OFF);
        }
        else if (ColorToggle == 5)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Tahuna Sands" BOLD_OFF);
        }
        else if (ColorToggle == 6)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Livid Brown" BOLD_OFF);
        }
        else if (ColorToggle == 7)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Neptune" BOLD_OFF);
        }
        else if (ColorToggle == 8)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Lochmara" BOLD_OFF);
        }
        else if (ColorToggle == 9)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Regal Blue" BOLD_OFF);
        }
        else if (ColorToggle == 10)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Vis Vis" BOLD_OFF);
        }
        else if (ColorToggle == 11)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Light Curious Blue" BOLD_OFF);
        }
        else if (ColorToggle == 12)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Ironside Grey" BOLD_OFF);
        }
        else if (ColorToggle == 13)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Yellow" BOLD_OFF);
        }
        else if (ColorToggle == 14)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Deco" BOLD_OFF);
        }
        else if (ColorToggle == 15)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Astronaut Blue" BOLD_OFF);
        }
         else if (ColorToggle == 16)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Bright Orange" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "DEFAULT" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf("m : set movement preset --> Current Preset:");
        if(NewBodyMovement == 0)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Random" BOLD_OFF);
        }
        else if(NewBodyMovement == 1)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Still" BOLD_OFF);
        }
        else if(NewBodyMovement == 2)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Sinusoidal" BOLD_OFF);
        }
        else if (NewBodyMovement == 3)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Circular" BOLD_OFF);
            if(selectCircleCenter)
            {
                printf("\n");
                printf("\033[0m");
                printf("Click to select circle center:");
            }
        
        }
        else if (NewBodyMovement == 4)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Oscillation" BOLD_OFF);

            printf("\n");
            printf("r/R: rotate the oscillation angle forwards/backwards\n");
            printf("+/-: increase/decrease the oscillation amplitude\n");
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "DEFAULT" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf(" i: Body Solidity On/Off Toggle --> ");
        if (!NewBodySolid)
        {
            printf("\033[0;31m");
            printf(BOLD_ON "Solid Off" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Solid On" BOLD_OFF);
        }
        
        printf("\n");
        printf("\033[0m");
        printf("e: Erase bodies toggle --> ");

        if (!EraseMode)
        {
            printf("\033[0;31m");
            printf(BOLD_ON "Off" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "On" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf("y/u: decrease/increase layer  --> Current Layer: %d", DrawLayer);

    }
    printf("\n");
}




int main(int argc, char** argv)
{
    setup();

    XWindowSize = 3000;
    YWindowSize = 1500;

    // Clip planes
    Near = 0.2;
    Far = 30.0;

    // Direction here your eye is located location
    EyeX = 0.0;
    EyeY = 0.0;
    EyeZ = 2.0;

    // Where you are looking
    CenterX = 0.0;
    CenterY = 0.0;
    CenterZ = 0.0;

    // Up vector for viewing
    UpX = 0.0;
    UpY = 1.0;
    UpZ = 0.0;

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
    glutInitWindowSize(XWindowSize, YWindowSize);
    glutInitWindowPosition(5, 5);
    Window = glutCreateWindow("N Body");

    gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
    glMatrixMode(GL_MODELVIEW);
    glClearColor(1.0, 1.0, 1.0, 1.0);
    loadBackgroundImage("../starry-king-of-the-monsters-hdtv.jpg");

    GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
    GLfloat light_ambient[] = {0.0, 0.0, 0.0, 1.0};
    GLfloat light_diffuse[] = {1.0, 1.0, 1.0, 1.0};
    GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
    GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
    GLfloat mat_specular[] = {1.0, 1.0, 1.0, 1.0};
    GLfloat mat_shininess[] = {10.0};
    glShadeModel(GL_SMOOTH);
    glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
    glLightfv(GL_LIGHT0, GL_POSITION, light_position);
    glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
    glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
    glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
    glDisable(GL_LIGHTING); //for lighting replace this and the next 2 lines with glEnable();
    glDisable(GL_LIGHT0);
    glDisable(GL_COLOR_MATERIAL);
    glEnable(GL_DEPTH_TEST);

    glutPassiveMotionFunc(mousePassiveMotionCallback);
    glutMouseFunc(mymouse);
    glutDisplayFunc(Display);
    glutReshapeFunc(reshape);
    glutKeyboardFunc(KeyPressed);
    glutIdleFunc(idle);
    terminalPrint();
    glutMainLoop();

    // Cleanup resources
    movieOff();
    freeBodies();

    return 0;
}





