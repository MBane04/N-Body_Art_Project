#include "hip/hip_runtime.h"
/*
    this file contains the following functions:
        void readBodiesFromFile(const char* filename);
        void writeBodiesToFile(const char* filename);
        void loadBackgroundImage(const char* filename);
*/

#include "./header.h"

void readBodiesFromFile(const char* filename)
{
    // Append the file directory to the file
    string fileDir = "../PreviousRuns/";
    fileDir.append(filename); // Now fileDir = "/PreviousRuns/filename"

    FILE* file = fopen(fileDir.c_str(), "r");
    if (file == NULL)
    {
        fprintf(stderr, "Error: Could not open file %s for reading\n", filename);
        return;
    }

    // Read the number of bodies from the top of the file
    int numBodiesFromFile;
    if (fscanf(file, "Number of bodies: %d\n", &numBodiesFromFile) != 1)
    {
        fprintf(stderr, "Error: Could not read the number of bodies from the file\n");
        fclose(file);
        return;
    }

    // Allocate memory based on the number of bodies
    capacity = numBodiesFromFile;
    bodies = (Body*)malloc(capacity * sizeof(Body));
    if (bodies == NULL)
    {
        fprintf(stderr, "Memory allocation failed\n");
        fclose(file);
        exit(1);
    }

    // Skip the header line
    char header[256];
    fgets(header, sizeof(header), file);

    // Read body information
    for (int i = 0; i < numBodiesFromFile; i++)
    {
        Body newBody;
        int isSolid;
        float color_x, color_y, color_z, color_w;
        float pos_x, pos_y, pos_z;
        float vel_x, vel_y, vel_z;
        float force_x, force_y, force_z;
        float initialX = 0.0f, initialY = 0.0f, oscillationAmplitude = 0.0f, oscillationAngle = 0.0f;
        int result = fscanf(file, "%d, %d, (%f, %f, %f, %f), %d, (%f, %f, %f), (%f, %f, %f), (%f, %f, %f), %f, %f, %f, %f, %f\n",
                            &newBody.id,
                            &isSolid,
                            &color_x, &color_y, &color_z, &color_w,
                            &newBody.movement,
                            &pos_x, &pos_y, &pos_z,
                            &vel_x, &vel_y, &vel_z,
                            &force_x, &force_y, &force_z,
                            &newBody.radius,
                            &initialX, &initialY, &oscillationAmplitude, &oscillationAngle);

        if (result == 17 || result == 21) // Old format or new format
        {
            newBody.isSolid = (bool)isSolid;
            newBody.color = make_float4(color_x, color_y, color_z, color_w);
            newBody.pos = make_float4(pos_x, pos_y, pos_z, 1.0f);
            newBody.vel = make_float4(vel_x, vel_y, vel_z, 0.0f);
            newBody.force = make_float4(force_x, force_y, force_z, 0.0f);

            if (result == 21) // New format
            {
                newBody.initialX = initialX;
                newBody.initialY = initialY;
                newBody.oscillationAmplitude = oscillationAmplitude;
                newBody.oscillationAngle = oscillationAngle;
            }
            else // Old format
            {
                newBody.initialX = 0.0f;
                newBody.initialY = 0.0f;
                newBody.oscillationAmplitude = 0.0f;
                newBody.oscillationAngle = 0.0f;
            }

            addBody(newBody);
        }
        else
        {
            fprintf(stderr, "Error: fscanf read %d values instead of 17 or 21\n", result);
            break;
        }
    }

    fclose(file);
    numBodies = numBodiesFromFile;
}

void writeBodiesToFile(const char* filename)
{
    string fileDir = "../PreviousRuns/";
    fileDir.append(filename); // Now fileDir = "/PreviousRuns/filename"

    FILE* file = fopen(fileDir.c_str(), "w");
    if (file == NULL)
    {
        fprintf(stderr, "Error: Could not open file %s for writing\n", filename);
        return;
    }

    // Write the number of bodies at the top of the file
    fprintf(file, "Number of bodies: %d\n", numBodies);

    // Write the header line
    fprintf(file, "ID, IsSolid, Color (R, G, B, A), Movement, Position (X, Y, Z), Velocity (X, Y, Z), Force (X, Y, Z), Radius, InitialX, InitialY, OscillationAmplitude, OscillationAngle\n");

    for (int i = 0; i < numBodies; i++)
    {
        fprintf(file, "%d, %d, (%f, %f, %f, %f), %d, (%f, %f, %f), (%f, %f, %f), (%f, %f, %f), %f, %f, %f, %f, %f\n",
                bodies[i].id,
                bodies[i].isSolid,
                bodies[i].color.x, bodies[i].color.y, bodies[i].color.z, bodies[i].color.w,
                bodies[i].movement,
                bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z,
                bodies[i].vel.x, bodies[i].vel.y, bodies[i].vel.z,
                bodies[i].force.x, bodies[i].force.y, bodies[i].force.z,
                bodies[i].radius,
                bodies[i].initialX, bodies[i].initialY, bodies[i].oscillationAmplitude, bodies[i].oscillationAngle);
    }

    fclose(file);
    printf("Body information written to %s\n", filename);
}

void loadBackgroundImage(const char* filename)
{
    printf("Attempting to load image: %s\n", filename); // Debug print
    
    backgroundTexture = SOIL_load_OGL_texture(
        filename,
        SOIL_LOAD_AUTO,
        SOIL_CREATE_NEW_ID,
        SOIL_FLAG_INVERT_Y
    );

    if (backgroundTexture == 0)
    {
        fprintf(stderr, "Error: Failed to load background image %s\n", filename);
        fprintf(stderr, "SOIL error: %s\n", SOIL_last_result()); // Print SOIL error message
    }
    else
    {
        printf("Successfully loaded background texture with ID: %u\n", backgroundTexture);
    }
}
