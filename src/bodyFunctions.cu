#include "hip/hip_runtime.h"
/*
    This file contains the following functions:
    float4 getColor(const char* colorName);
    void screenToWorld(int x, int y, float* worldX, float* worldY);
    void addBody(Body newBody);
    void addBodyAtPosition(float x, float y);
    void removeBodyAtPosition(float x, float y);
    void freeBodies();


*/

#include "./header.h"

// Helper functions
float4 getColor(const char* colorName) {
    if (strcmp(colorName, "paris_m") == 0) return colors.paris_m;
    if (strcmp(colorName, "manz") == 0) return colors.manz;
    if (strcmp(colorName, "outer_space") == 0) return colors.outer_space;
    if (strcmp(colorName, "curious_blue") == 0) return colors.curious_blue;
    if (strcmp(colorName, "tahuna_sands") == 0) return colors.tahuna_sands;
    if (strcmp(colorName, "livid_brown") == 0) return colors.livid_brown;
    if (strcmp(colorName, "neptune") == 0) return colors.neptune;
    if (strcmp(colorName, "lochmara") == 0) return colors.lochmara;
    if (strcmp(colorName, "regal_blue") == 0) return colors.regal_blue;
    if (strcmp(colorName, "vis_vis") == 0) return colors.vis_vis;
    if (strcmp(colorName, "light_curious_blue") == 0) return colors.light_curious_blue;
    if (strcmp(colorName, "ironside_grey") == 0) return colors.ironside_grey;
    if (strcmp(colorName, "yellow") == 0) return colors.yellow;
    if (strcmp(colorName, "deco") == 0) return colors.deco;
    if (strcmp(colorName, "astronaut_blue") == 0) return colors.astronaut_blue;
    if (strcmp(colorName, "bright_orange") == 0) return colors.bright_orange;
    return (float4){0.0, 0.0, 0.0, 1.0}; // Default value
}

void screenToWorld(int x, int y, float* worldX, float* worldY) {
    *worldX = (5.76 * x / XWindowSize) - 1.84f; // Map x to (-1.84, 1.84)
    *worldY = -(2.9f * y / YWindowSize) + 1.0f;   // Map y to (-1, 1)
    //printf("Converted screen (%d, %d) to world (%f, %f)\n", x, y, *worldX, *worldY); // Debugging statement
}

// Body Management functions
void addBody(Body newBody) {
    // Reallocate memory to accommodate the new body
    if (numBodies >= capacity) {
        capacity *= 2; //double the capacity
        Body* temp = (Body*)realloc(bodies, capacity * sizeof(Body));
        if (temp == NULL) {
            fprintf(stderr, "Memory allocation failed\n");
            exit(1);
        }
        bodies = temp; //assign the new memory to the bodies array
    }

    // Set movement-specific properties
    if(newBody.movement == 0) { //random movement
        newBody.vel.x = ((float)rand()/(float)RAND_MAX)*2.0f - 1.0f;
        newBody.vel.y = ((float)rand()/(float)RAND_MAX)*2.0f - 1.0f;
        newBody.vel.z = 0.0;
    }
    else if (newBody.movement == 1) { //still
        newBody.vel.x = 0.0f;
        newBody.vel.y = 0.0f;
        newBody.vel.z = 0.0f;
    }
    else if (newBody.movement == 2) { //sinusoidal
        newBody.vel.x = 0.2f;
        newBody.vel.y = 0.0f;
        newBody.vel.z = 0.0f;
        newBody.initialY = newBody.pos.y; // Store the initial y position
    }
    else if (newBody.movement == 3) { //circular
        newBody.circle.x = circleCenterX; // Store the center x position
        newBody.circle.y = circleCenterY; // Store the center y position
        newBody.circle.z = atan2(newBody.pos.y - circleCenterY, newBody.pos.x - circleCenterX); // Calculate the initial angle
        newBody.circle.w = sqrt(pow(newBody.pos.x - circleCenterX, 2) + pow(newBody.pos.y - circleCenterY, 2)); // Calculate the radius
    }
    else if (newBody.movement == 4) { //oscillating
        newBody.initialX = newBody.pos.x;
        newBody.initialY = newBody.pos.y;
        newBody.oscillationAmplitude = currentOscillationAmplitude; 
        newBody.oscillationAngle = currentOscillationAngle;
    }

    // Add the new body to the array
    bodies[numBodies] = newBody;
    
    // Increment the number of bodies
    numBodies++;
}

void addBodyAtPosition(float x, float y) {
    Body newBody;

    // Set the color based on the ColorToggle value
    switch (ColorToggle) {
        case 1: newBody.color = getColor("paris_m"); break;
        case 2: newBody.color = getColor("manz"); break;
        case 3: newBody.color = getColor("outer_space"); break;
        case 4: newBody.color = getColor("curious_blue"); break;
        case 5: newBody.color = getColor("tahuna_sands"); break;
        case 6: newBody.color = getColor("livid_brown"); break;
        case 7: newBody.color = getColor("neptune"); break;
        case 8: newBody.color = getColor("lochmara"); break;
        case 9: newBody.color = getColor("regal_blue"); break;
        case 10: newBody.color = getColor("vis_vis"); break;
        case 11: newBody.color = getColor("light_curious_blue"); break;
        case 12: newBody.color = getColor("ironside_grey"); break;
        case 13: newBody.color = getColor("yellow"); break;
        case 14: newBody.color = getColor("deco"); break;
        case 15: newBody.color = getColor("astronaut_blue"); break;
        default: newBody.color = getColor("default"); break;
    }

    //set remaining properties of the new body
    newBody.id = numBodies;
    newBody.pos = make_float4(x, y, 0.0 + DrawLayer / 100.0, 1.0f);
    newBody.vel = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    newBody.force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    newBody.radius = newBodyRadius * DiameterOfBody / 2.0f;
    newBody.isSolid = NewBodySolid;
    newBody.movement = NewBodyMovement;

    if (NewBodyMovement == 4) { // Oscillation movement
        newBody.initialX = x;
        newBody.initialY = y;
        newBody.oscillationAmplitude = currentOscillationAmplitude;
        newBody.oscillationAngle = currentOscillationAngle;
    }

    addBody(newBody);
}

void removeBodyAtPosition(float x, float y) {
    for (int i = 0; i < numBodies; ++i) {
        float dx = bodies[i].pos.x - x;
        float dy = bodies[i].pos.y - y;
        float distance = sqrt(dx * dx + dy * dy);

        if (distance < bodies[i].radius) {
            // Remove the body by shifting the remaining bodies
            for (int j = i; j < numBodies - 1; ++j) {
                bodies[j] = bodies[j + 1];
            }
            --numBodies;
            return;
        }
    }
}

void freeBodies() {
    free(bodies);
    bodies = NULL;
}