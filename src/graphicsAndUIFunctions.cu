#include "hip/hip_runtime.h"
/*
    This file contains the following functions:
        void drawPicture();
        void drawGrid(float spacing, int numLines);
        void display();
        void renderBackground();
        void terminalPrint();
*/

#include "./header.h"


#include "./header.h"
#include <vector>

// Render a sphere with given radius, latitude segments, and longitude segments
void renderSphere(float radius, int slices, int stacks) //replaces glutSolidSphere
{
    // Generate sphere vertices
    std::vector<float> vertices;
    std::vector<float> normals;
    std::vector<unsigned int> indices;
    
    for (int lat = 0; lat <= slices; lat++) {
        float theta = lat * M_PI / slices;
        float sinTheta = sin(theta);
        float cosTheta = cos(theta);
        
        for (int lon = 0; lon <= stacks; lon++) {
            float phi = lon * 2 * M_PI / stacks;
            float sinPhi = sin(phi);
            float cosPhi = cos(phi);
            
            float x = cosPhi * sinTheta;
            float y = cosTheta;
            float z = sinPhi * sinTheta;
            
            normals.push_back(x);
            normals.push_back(y);
            normals.push_back(z);
            
            vertices.push_back(radius * x);
            vertices.push_back(radius * y);
            vertices.push_back(radius * z);
        }
    }
    
    // Generate indices
    for (int lat = 0; lat < slices; lat++) 
    {
        for (int lon = 0; lon < stacks; lon++) 
        {
            int first = (lat * (stacks + 1)) + lon;
            int second = first + stacks + 1;
            
            // Draw a quad for each segment
            glBegin(GL_QUADS);
            
            glNormal3f(normals[first * 3], normals[first * 3 + 1], normals[first * 3 + 2]);
            glVertex3f(vertices[first * 3], vertices[first * 3 + 1], vertices[first * 3 + 2]);
            
            glNormal3f(normals[first * 3 + 3], normals[first * 3 + 4], normals[first * 3 + 5]);
            glVertex3f(vertices[first * 3 + 3], vertices[first * 3 + 4], vertices[first * 3 + 5]);
            
            glNormal3f(normals[second * 3 + 3], normals[second * 3 + 4], normals[second * 3 + 5]);
            glVertex3f(vertices[second * 3 + 3], vertices[second * 3 + 4], vertices[second * 3 + 5]);
            
            glNormal3f(normals[second * 3], normals[second * 3 + 1], normals[second * 3 + 2]);
            glVertex3f(vertices[second * 3], vertices[second * 3 + 1], vertices[second * 3 + 2]);
            
            glEnd();
        }
    }
}

void drawPicture()
{
    if (Trace == 0)
    {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    }

    // Render the background image
    if (BackgroundToggle && backgroundTexture != 0)
    {
        renderBackground();
    }

    if (GridOn) drawGrid(0.5f, 50);  // Wider grid spacing, more grid lines

    if (NewBodyToggle == 1)
    {
        float4 mouseColor;

        if (ColorToggle == 1)
        {
            //color paris m
            mouseColor = getColor("paris_m");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 2)
        {
            //color manz
            mouseColor = getColor("manz");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 3)
        {
            //color outer space
            mouseColor = getColor("outer_space");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 4)
        {
            //color curious blue
            mouseColor = getColor("curious_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 5)
        {
            //color tahuna sands
            mouseColor = getColor("tahuna_sands");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 6)
        {
            //color livid brown
            mouseColor = getColor("livid_brown");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 7)
        {
            //color neptune
            mouseColor = getColor("neptune");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 8)
        {
            //color lochmara
            mouseColor = getColor("lochmara");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 9)
        {
            //color regal blue
            mouseColor = getColor("regal_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 10)
        {
            //color vis vis
            mouseColor = getColor("vis_vis");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 11)
        {
            //color light curious blue
            mouseColor = getColor("light_curious_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 12)
        {
            //color ironside grey
            mouseColor = getColor("ironside_grey");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 13)
        {
            //color yellow
            mouseColor = getColor("yellow");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 14)
        {
            //color deco
            mouseColor = getColor("deco");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 15)
        {
            //color astronaut blue
            mouseColor = getColor("astronaut_blue");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else if (ColorToggle == 16)
        {
            //color astronaut blue
            mouseColor = getColor("bright_orange");
            glColor3d(mouseColor.x, mouseColor.y, mouseColor.z);
        }
        else
        {
            //color white
            glColor3d(1.0, 1.0, 1.0);
        }
        glPushMatrix();
        glTranslatef(MouseX, MouseY, MouseZ + DrawLayer / 100.0f);
        renderSphere(newBodyRadius * DiameterOfBody / 2.0, 20, 20);
        glPopMatrix();

        // Draw the oscillation path line
        if (NewBodyMovement == 4)
        {
            float dx = currentOscillationAmplitude * cos(currentOscillationAngle);
            float dy = currentOscillationAmplitude * sin(currentOscillationAngle);
            glColor3f(1.0f, 0.0f, 0.0f); // Red color for the line
            glBegin(GL_LINES);
            // Draw line in front of the body
            glVertex3f(MouseX, MouseY, MouseZ);
            glVertex3f(MouseX + dx, MouseY + dy, MouseZ);
            // Draw line behind the body
            glVertex3f(MouseX, MouseY, MouseZ);
            glVertex3f(MouseX - dx, MouseY - dy, MouseZ);
            glEnd();
        }
    }

    for (int i = 0; i < numBodies; i++)
    {
        glColor3d(bodies[i].color.x, bodies[i].color.y, bodies[i].color.z);
        glPushMatrix();
        glTranslatef(bodies[i].pos.x, bodies[i].pos.y, bodies[i].pos.z);
        //glutSolidSphere(bodies[i].radius, 20, 20);, args are radius, slices (vertical) , stacks (horizontal)
        renderSphere(bodies[i].radius, 20, 20);
        glPopMatrix();
    }

    glfwSwapBuffers(window); //changed from glutSwapBuffers();

    if (MovieOn == 1)
    {
        glReadPixels(0, 0, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, Buffer);
        fwrite(Buffer, sizeof(int) * XWindowSize * YWindowSize, 1, MovieFile);
    }

 
}

void drawGrid(float spacing, int numLines)
{
    glColor3f(0.8f, 0.8f, 0.8f); // Set grid color (light gray)
    glBegin(GL_LINES);

    // Draw vertical lines
    for (int i = -numLines; i <= numLines; ++i)
    {
        float x = i * spacing;
        glVertex3f(x, -numLines * spacing, 0.0f);
        glVertex3f(x, numLines * spacing, 0.0f);
    }

    // Draw horizontal lines
    for (int i = -numLines; i <= numLines; ++i)
    {
        float y = i * spacing;
        glVertex3f(-numLines * spacing, y, 0.0f);
        glVertex3f(numLines * spacing, y, 0.0f);
    }

    glEnd();
}


void display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    renderBackground();
	drawPicture();
}

void renderBackground()
{
    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, backgroundTexture);

    // Save the current color state
    GLboolean colorMask[4];
    glGetBooleanv(GL_COLOR_WRITEMASK, colorMask);
    GLfloat currentColor[4];
    glGetFloatv(GL_CURRENT_COLOR, currentColor);

    // Reset color to white
    glColor3f(1.0f, 1.0f, 1.0f);

    // Calculate aspect ratio
    float windowAspect = (float)XWindowSize / (float)YWindowSize;

    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex3f(-windowAspect, -1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex3f(windowAspect, -1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex3f(windowAspect, 1.0f, -1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex3f(-windowAspect, 1.0f, -1.0f);
    glEnd();

    glDisable(GL_TEXTURE_2D);

    // Restore the previous color state
    glColor4fv(currentColor);
    glColorMask(colorMask[0], colorMask[1], colorMask[2], colorMask[3]);
}

void terminalPrint()
{
	/*
	default  \033[0m
	Black:   \033[0;30m
	Red:     \033[0;31m
	Green:   \033[0;32m
	Yellow:  \033[0;33m
	Blue:    \033[0;34m
	Magenta: \033[0;35m
	Cyan:    \033[0;36m
	White:   \033[0;37m
	printf("\033[0;30mThis text is black.\033[0m\n");
	
	BOLD_ON  "\e[1m"
	BOLD_OFF   "\e[m"
	*/
	
	system("clear");
	
    printf("\n");
	printf("\n S: Screenshot");
	
	printf("\n");
	printf("\n q: Terminates the simulation");

    printf("\n");
    printf("\n s: Save this run");

	printf("\n\n");
	printf("\033[0m");
	printf(" p: Pause on/off toggle --> ");
	printf(" The simulation is:");
	if (Pause == 1) 
	{
		printf("\e[1m" " \033[0;31mPaused\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mRunning\n" "\e[m");
	}
	
	printf("\n");
	printf("\033[0m");
	printf(" t: Trace on/off toggle --> ");
	printf(" Trace is:");
	if (Trace == 1) 
	{
		printf("\e[1m" " \033[0;31mOn\n" "\e[m");
	}
	else 
	{
		printf("\e[1m" " \033[0;32mOff\n" "\e[m");
	}
	//printf("\n");
	//printf("\033[0m");
	//printf(" v: Toggle view (Perspective/Orthogonal) --> ");
	//printf(" Current View: ");
	// if (isOrthogonal) 
	// {
	// 	printf("\e[1m" " \033[0;32mOrthogonal\n" "\e[m");
	// }
	// else 
	// {
	// 	printf("\e[1m" " \033[0;31mDefault\n" "\e[m");
	// }
	printf("\n M: Video On/Off toggle --> ");
	if (MovieFlag == 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "Video Recording Off\n" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Video Recording On\n" BOLD_OFF);
	}
    printf("\n");
    printf("\033[0m");
    printf(" g: Grid On/Off Toggle --> ");
    if (GridOn)
    {
        printf("\033[0;32m");
        printf(BOLD_ON "Grid On" BOLD_OFF);
    }
    else
    {
        printf("\033[0;31m");
        printf(BOLD_ON "Grid Off" BOLD_OFF);
    }

    printf("\n");
    printf("\033[0m");
    printf(" b: Background On/Off Toggle --> ");
    if (BackgroundToggle)
    {
        printf("\033[0;32m");
        printf(BOLD_ON "Background On" BOLD_OFF);
    }
    else
    {
        printf("\033[0;31m");
        printf(BOLD_ON "Background Off" BOLD_OFF);
    }



	printf("\n n: Simulaton Mode Add View/Add Body Toggle --> Mode:");
	if (NewBodyToggle== 0) 
	{
		printf("\033[0;31m");
		printf(BOLD_ON "View" BOLD_OFF); 
	}
	else 
	{
		printf("\033[0;32m");
		printf(BOLD_ON "Add Body" BOLD_OFF);
	}
	//controls for body placement
    if(NewBodyToggle == 1)
    {
        printf("\n");
        printf("\033[0m");
        printf(" [/]: Change radius of new body backwards/forwards\n");

        printf("\n");
        printf("\033[0m");
        printf(" k/l: Change color of new body backwards/forwards\n");
        printf(" Current Color: ");
        if (ColorToggle == 1)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Paris M" BOLD_OFF);
        }
        else if (ColorToggle == 2)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Manz" BOLD_OFF);
        }
        else if (ColorToggle == 3)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Outer Space" BOLD_OFF);
        }
        else if (ColorToggle == 4)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Curious Blue" BOLD_OFF);
        }
        else if (ColorToggle == 5)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Tahuna Sands" BOLD_OFF);
        }
        else if (ColorToggle == 6)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Livid Brown" BOLD_OFF);
        }
        else if (ColorToggle == 7)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Neptune" BOLD_OFF);
        }
        else if (ColorToggle == 8)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Lochmara" BOLD_OFF);
        }
        else if (ColorToggle == 9)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Regal Blue" BOLD_OFF);
        }
        else if (ColorToggle == 10)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Vis Vis" BOLD_OFF);
        }
        else if (ColorToggle == 11)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Light Curious Blue" BOLD_OFF);
        }
        else if (ColorToggle == 12)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Ironside Grey" BOLD_OFF);
        }
        else if (ColorToggle == 13)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Yellow" BOLD_OFF);
        }
        else if (ColorToggle == 14)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Deco" BOLD_OFF);
        }
        else if (ColorToggle == 15)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Astronaut Blue" BOLD_OFF);
        }
         else if (ColorToggle == 16)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Bright Orange" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "DEFAULT" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf("m : set movement preset --> Current Preset:");
        if(NewBodyMovement == 0)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Random" BOLD_OFF);
        }
        else if(NewBodyMovement == 1)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Still" BOLD_OFF);
        }
        else if(NewBodyMovement == 2)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Sinusoidal" BOLD_OFF);
        }
        else if (NewBodyMovement == 3)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Circular" BOLD_OFF);
            if(selectCircleCenter)
            {
                printf("\n");
                printf("\033[0m");
                printf("Click to select circle center:");
            }
        
        }
        else if (NewBodyMovement == 4)
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Oscillation" BOLD_OFF);

            printf("\n");
            printf("r/R: rotate the oscillation angle forwards/backwards\n");
            printf("+/-: increase/decrease the oscillation amplitude\n");
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "DEFAULT" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf(" i: Body Solidity On/Off Toggle --> ");
        if (!NewBodySolid)
        {
            printf("\033[0;31m");
            printf(BOLD_ON "Solid Off" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "Solid On" BOLD_OFF);
        }
        
        printf("\n");
        printf("\033[0m");
        printf("e: Erase bodies toggle --> ");

        if (!EraseMode)
        {
            printf("\033[0;31m");
            printf(BOLD_ON "Off" BOLD_OFF);
        }
        else
        {
            printf("\033[0;32m");
            printf(BOLD_ON "On" BOLD_OFF);
        }

        printf("\n");
        printf("\033[0m");
        printf("y/u: decrease/increase layer  --> Current Layer: %d", DrawLayer);

    }
    printf("\n");
}