#include "hip/hip_runtime.h"
#include "./header.h"

/*
    this file contains the following functions:

        void cursor_position_adapter(GLFWwindow* window, double xpos, double ypos);
        void mouse_button_adapter(GLFWwindow* window, int button, int action, int mods);
        void key_adapter(GLFWwindow* window, int key, int scancode, int action, int mods);
        void idle();
        void reshape(int w, int h);
        void keyPressed(unsigned char key, int x, int y);
        void mousePassiveMotionCallback(int x, int y);
        void myMouse(int button, int state, int x, int y);
        string getTimeStamp();
        void handle_sigpipe(int sig);
        void movieOn();
        void movieOff();
        void screenShot();
        
*/

//Adapts cursor from GLFW to GLUT so we don't have to change the code
void cursor_position_adapter(GLFWwindow* window, double xpos, double ypos)
{
    mousePassiveMotionCallback((int)xpos, (int)ypos);
}

//Adapts mouse button from GLFW to GLUT so we don't have to change the code
void mouse_button_adapter(GLFWwindow* window, int button, int action, int mods)
{
    int glutButton = (button == GLFW_MOUSE_BUTTON_LEFT) ? 0 : 
                    (button == GLFW_MOUSE_BUTTON_MIDDLE) ? 1 : 2;
    int glutState = (action == GLFW_PRESS) ? 0 : 1;
    
    double xpos, ypos;
    glfwGetCursorPos(window, &xpos, &ypos);
    
    myMouse(glutButton, glutState, (int)xpos, (int)ypos);
}

//Adapts key press from GLFW to GLUT so we don't have to change the code
//Adapts key press from GLFW to GLUT so we don't have to change the code
void key_adapter(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    printf("Key pressed: %d, action: %d, mods: %d\n", key, action, mods);
    if (action == GLFW_PRESS || action == GLFW_REPEAT) 
    {
        // Handle letter keys (A-Z)
        if (key >= GLFW_KEY_A && key <= GLFW_KEY_Z)
        {
            // Convert to the correct case based on shift modifier
            unsigned char charKey;
            if (mods & GLFW_MOD_SHIFT)
                charKey = (unsigned char)key; // Keep uppercase (65-90)
            else
                charKey = (unsigned char)(key + 32); // Convert to lowercase (97-122)
            
            printf("Converted key: %c (%d)\n", charKey, charKey);
            keyPressed(charKey, 0, 0);
        }
        // Handle other printable ASCII characters
        else if (key >= 32 && key <= 126)
        {
            printf("ASCII key: %c (%d)\n", (unsigned char)key, key);
            keyPressed((unsigned char)key, 0, 0);
        }
        else if (key == GLFW_KEY_ESCAPE)
        {
            glfwSetWindowShouldClose(window, GLFW_TRUE);
        }
        // Add special key handling here if needed
        // else if (key == GLFW_KEY_UP) { ... }
    }
}

void idle()
{
    if (NewBodyToggle == 1)
    {
        drawPicture();
    }
    else
    {
        nBody();
    }
}

// Update reshape to work with GLFW
void reshape(GLFWwindow* window, int w, int h)
{
    // Prevent division by zero
    if (h == 0) h = 1;

    // Update global window size variables
    XWindowSize = w;
    YWindowSize = h;

    // Calculate the aspect ratio of the window
    float aspectRatio = (float)w / (float)h; 

    // Set the viewport to cover the new window
    glViewport(0, 0, (GLsizei)w, (GLsizei)h);

    // Set the projection matrix
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    // Adjust the projection matrix to maintain the aspect ratio of the bodies
    if (isOrthogonal) 
    {
        if (aspectRatio >= 1.0f) 
        {
            // Window is wider than it is tall
            glOrtho(-1.0 * aspectRatio, 1.0 * aspectRatio, -1.0, 1.0, Near, Far);
        } 
        else 
        {
            // Window is taller than it is wide
            glOrtho(-1.0, 1.0, -1.0 / aspectRatio, 1.0 / aspectRatio, Near, Far);
        }
    } 
    else 
    {
        if (aspectRatio >= 1.0f) 
        {
            // Window is wider than it is tall
            glFrustum(-0.2 * aspectRatio, 0.2 * aspectRatio, -0.2, 0.2, Near, Far);
        } else 
        {
            // Window is taller than it is wide
            glFrustum(-0.2, 0.2, -0.2 / aspectRatio, 0.2 / aspectRatio, Near, Far);
        }
    }

    // Switch back to the modelview matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
}

void keyPressed(unsigned char key, int x, int y)
{	
	if(key == 'q')
	{
		// Check if ffmpeg is not NULL before closing
        if (ffmpeg != NULL) 
		{
            pclose(ffmpeg);
            ffmpeg = NULL; // Optionally set to NULL after closing
        } 
		else 
		{
            fprintf(stderr, "Warning: Attempted to close a NULL file pointer\n");
        }
        // Replace glutDestroyWindow(Window) with:
        glfwSetWindowShouldClose(window, GLFW_TRUE);
        printf("\nw Good Bye\n");
        return; //return, not exit so glfw can clean up
	}
	// if(key == 'v') //not much need for this anymore
   	// {
    //     // Toggle the view mode
    //     isOrthogonal = !isOrthogonal;

    //     // Call reshape to update the projection matrix
    //     reshape(glutGet(GLUT_WINDOW_WIDTH), glutGet(GLUT_WINDOW_HEIGHT));

    //     // Redraw the scene
    //     glutPostRedisplay();
   	//  }
	if(key == 'p')
	{
		if(Pause == 1) Pause = 0;
		else Pause = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 't') // Turns tracers on and off
	{
		if(Trace == 1) Trace = 0;
		else Trace = 1;
		drawPicture();
		terminalPrint();
	}
	if(key == 'M')  // Movie on/off
	{
		if(MovieFlag == 0) 
		{
			MovieFlag = 1;
			movieOn();
		}
		else 
		{
			MovieFlag = 0;
			movieOff();
		}
		terminalPrint();
	}
	
	if(key == 'S')  // Screenshot
	{	
		screenShot();
		terminalPrint();
	}
	if (key == 'n') // Add a new body
	{
		if(NewBodyToggle == 0) NewBodyToggle = 1;
		else NewBodyToggle = 0;
		terminalPrint();
	}
	if(key == ']')  
	{
		newBodyRadius += 0.01;
		terminalPrint();
		//printf("\n Your selection area = %f times the radius of atrium. \n", HitMultiplier);
	}
	if(key == '[')
	{
		newBodyRadius -= 0.01;
		if(newBodyRadius < 0.0) newBodyRadius = 0.0;
		terminalPrint();
		//printf("\n Your selection area = %f times the radius of atrium. \n", HitMultiplier);
	}
	if(key == 's')
	{
        printf("Enter the file name to save this run to: ");
        char filename[256];
        scanf("%s", filename);
        writeBodiesToFile(filename);
	}
    if(key == 'e')
    {
        if(EraseMode)
        {
            EraseMode = false;
            terminalPrint();
        }
        else
        {
            EraseMode = true;
            terminalPrint();
        }
    }
    if(key == 'g')
    {
        if(GridOn)
        {
            GridOn = false;
            drawPicture();
            terminalPrint();
        }
        else
        {
            GridOn = true;
            drawPicture();
            terminalPrint();
        }
    }

    if(key == 'b')
    {
        if(BackgroundToggle >= 0 && BackgroundToggle < 2)
        {
            BackgroundToggle++;
        }
        else
        {
            BackgroundToggle = 0;
        }
        if(BackgroundToggle == 1)
        {
            loadBackgroundImage("../starry-king-of-the-monsters-hdtv.jpg");
            drawPicture();
            terminalPrint();
        }
        else if (BackgroundToggle == 2)
        {
            loadBackgroundImage("../godzilla_background7.png");
            drawPicture();
            terminalPrint();
        }
        else
        {
            BackgroundToggle = 0;
            drawPicture();
            terminalPrint();
        }
    }

    if(NewBodyToggle == 1)
    {
        if (key == 'l') // cycle through colors, forward
        {
            if (ColorToggle < 16)
            {
                ColorToggle++;
            }
            else
            {
                ColorToggle = 1;
            }
            terminalPrint();
        }
        if (key == 'k') // cycle through colors, backward
        {
            if (ColorToggle > 1)
            {
                ColorToggle--;
            }
            else
            {
                ColorToggle = 15;
            }
            terminalPrint();
        }
        //set movement pattern
        //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!CHANGES NEED TO BE MADE LATER!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        if (key == 'm')
        {
            printf("Enter the movement pattern for the new body: ");
            printf("0 for random movement, 1 for still, 2 for sinusoidal, 3 for circular, 4 for oscillating\n");
            scanf("%d", &NewBodyMovement);
            if (NewBodyMovement < 0 || NewBodyMovement > 4)
            {
                printf("Invalid movement pattern.\n");
                NewBodyMovement = 0;
            }
            if(NewBodyMovement == 3)
            {
                //get the center of the circle from the user using the mouse
                selectCircleCenter = true;
                currentOscillationAmplitude = 0.0f;
            }
            
            terminalPrint();
        }

        if(key == 'i')//is the new body solid?
        {
            if(NewBodySolid == true)
            {
                NewBodySolid = false;
            }
            else
            {
                NewBodySolid = true;
            }
            terminalPrint();
        }

        //add DrawLayer so you can decide what appears on top of what
        if(key == 'u')
        {
            DrawLayer++;
            drawPicture();
            terminalPrint();
        }
        if(key == 'y')
        {
            DrawLayer--;
            drawPicture();
            terminalPrint();
        }

       if(NewBodyMovement == 4)
       {
            if (key == 'r') // Rotate oscillation angle left
            {
                currentOscillationAngle -= 0.1f; // Adjust the angle increment as needed
                if (currentOscillationAngle < 0.0f)
                {
                    currentOscillationAngle += 2.0f * M_PI;
                }
                drawPicture();
                terminalPrint();
            }
            if (key == 'R') // Rotate oscillation angle right
            {
                currentOscillationAngle += 0.1f; // Adjust the angle increment as needed
                if (currentOscillationAngle >= 2.0f * M_PI)
                {
                    currentOscillationAngle -= 2.0f * M_PI;
                }
                drawPicture();
                terminalPrint();
            }

            if (key == '+') // Increase oscillation amplitude
            {
                currentOscillationAmplitude += 0.01f; // Adjust the amplitude increment as needed
                drawPicture();
                terminalPrint();
            }
            if (key == '-') // Decrease oscillation amplitude
            {
                currentOscillationAmplitude -= 0.01f; // Adjust the amplitude increment as needed
                if (currentOscillationAmplitude < 0.0f)
                {
                    currentOscillationAmplitude = 0.0f;
                }
                drawPicture();
                terminalPrint();
            }
       }
    }
}

void mousePassiveMotionCallback(int x, int y) 
{
    // Use proper conversion function instead of magic numbers
    float worldX, worldY;
    screenToWorld(x, y, &worldX, &worldY);
    
    MouseX = worldX;
    MouseY = worldY;
    MouseZ = 0.0f;
    
    if (IsDragging)
    {
        if(EraseMode)
            removeBodyAtPosition(MouseX, MouseY);
        else
            addBodyAtPosition(MouseX, MouseY);
    }
}

// This is called when you push a mouse button.
void myMouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{	
		if(button == GLUT_LEFT_BUTTON)
		{	
			if(NewBodyToggle == 1)
			{
                if(EraseMode)
                {
                    removeBodyAtPosition(MouseX, MouseY);
                }
                else if(selectCircleCenter)
                {
                    // Convert screen coordinates to world coordinates
                    screenToWorld(x, y, &circleCenterX, &circleCenterY);
                    printf("Circle center selected at (%f, %f)\n", circleCenterX, circleCenterY);
                    selectCircleCenter = false; // Reset the flag
                }
                else
                {
                    //generate random numbers for all the properties of the new body
                    
                    int index = numBodies; // Define and initialize index

                    // Convert window coordinates to OpenGL coordinates
                    screenToWorld(x, y, &MouseX, &MouseY);
                    MouseZ = 0.0f + DrawLayer/100.0f; // Keep the Z offset for drawing layers

                    // Print the converted coordinates for debugging
                    printf("MouseX: %f, MouseY: %f, MouseZ: %f\n", MouseX, MouseY, MouseZ);

                    Body newBody; //create a new body with the body struct

                    // Set the color of the new body based on the ColorToggle

                    if(ColorToggle == 1)
                    {
                        newBody.color = getColor("paris_m");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 2)
                    {
                        newBody.color = getColor("manz");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 3)
                    {
                        newBody.color = getColor("outer_space");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 4)
                    {
                        newBody.color = getColor("curious_blue");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 5)
                    {
                        newBody.color = getColor("tahuna_sands");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 6)
                    {
                        newBody.color = getColor("livid_brown");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 7)
                    {
                        newBody.color = getColor("neptune");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 8)
                    {
                        newBody.color = getColor("lochmara");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 9)
                    {
                        newBody.color = getColor("regal_blue");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 10)
                    {
                        newBody.color = getColor("vis_vis");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 11)
                    {
                        newBody.color = getColor("light_curious_blue");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 12)
                    {
                        newBody.color = getColor("ironside_grey");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 13)
                    {
                        newBody.color = getColor("yellow");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 14)
                    {
                        newBody.color = getColor("deco");
                        HotkeyPrint = 0;
                    }
                    else if(ColorToggle == 15)
                    {
                        newBody.color = getColor("astronaut_blue");
                        HotkeyPrint = 0;
                    }
                      else if(ColorToggle == 16)
                    {
                        newBody.color = getColor("bright_orange");
                        HotkeyPrint = 0;
                    }
                    else
                    {
                        newBody.color =  {1.0f, 1.0f, 1.0f, 1.0f}; //default
                    }

                    //assign all the properties of the new body
                    newBody.id = index;
                    newBody.isSolid = true;
                    newBody.movement = NewBodyMovement;
                    newBody.pos = {MouseX, MouseY, MouseZ + DrawLayer/100.0f, 1.0f}; // Directly assign values to float4
                    newBody.force = {0.0f, 0.0f, 0.0f, 0.0f}; // Directly assign values to float4
                    newBody.radius = newBodyRadius * DiameterOfBody/2.0f;

                    addBody(newBody);
                }   
            }
		}
		else if(button == GLUT_RIGHT_BUTTON) // Right Mouse button down
		{
            if (state == GLUT_DOWN)
            {
                //make it a toggle
                if(IsDragging == false)
                {
                    IsDragging = true;
                    screenToWorld(x, y, &MouseX, &MouseY);
                    MouseZ = 0.0f + DrawLayer/100.0f; // Keep the Z offset for drawing layers
                }
                else
                {
                    IsDragging = false;
                }
            }
            else if (state == GLUT_UP)
            {
                IsDragging = false;
                printf("Mouse up at (%f, %f)\n", MouseX, MouseY); // Debugging statement
            }
		}
		else if(button == GLUT_MIDDLE_BUTTON)
		{
			// Do stuff in here if you choose to when the middle mouse button is pressed.
		}
	}
	
	// If no mouse button is down (state 0, they don't have a nice word like GLUT_NOT_DOWN) 
	// but you move the mouse wheel this is called.
	if(state == 0)
	{
		// When you turn the mouse whell forward this is called.
		if(button == 3)
		{
			EyeZ -=0.1;
			
		}
		
		// When you turn the mouse whell backward this is called.
		else if(button == 4)
		{
			EyeZ += 0.1;
			
		}
	}
	glLoadIdentity();
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	//glutPostRedisplay();
}

string getTimeStamp()
{
	// Want to get a time stamp string representing current date/time, so we have a
	// unique name for each video/screenshot taken.
	time_t t = time(0); 
	struct tm * now = localtime( & t );
	int month = now->tm_mon + 1, day = now->tm_mday, year = now->tm_year, 
				curTimeHour = now->tm_hour, curTimeMin = now->tm_min, curTimeSec = now->tm_sec;
	stringstream smonth, sday, syear, stimeHour, stimeMin, stimeSec;
	smonth << month;
	sday << day;
	syear << (year + 1900); // The computer starts counting from the year 1900, so 1900 is year 0. So we fix that.
	stimeHour << curTimeHour;
	stimeMin << curTimeMin;
	stimeSec << curTimeSec;
	string timeStamp;
	if (curTimeMin <= 9)	
		timeStamp = smonth.str() + "-" + sday.str() + "-" + syear.str() + '_' + stimeHour.str() + ".0" + stimeMin.str() + 
					"." + stimeSec.str();
	else			
		timeStamp = smonth.str() + "-" + sday.str() + '-' + syear.str() + "_" + stimeHour.str() + "." + stimeMin.str() +
					"." + stimeSec.str();
	return timeStamp;
}

// Signal handler for SIGPIPE
void handle_sigpipe(int sig)
{
    fprintf(stderr, "Caught SIGPIPE signal: %d\n", sig);
}

void movieOn()
{
    // Register the SIGPIPE signal handler
    signal(SIGPIPE, handle_sigpipe);

    string ts = getTimeStamp();
    ts.append(".mp4");

    // Convert the x and y window size to a string of format "XsizexYsize"
    stringstream ss;
    ss << XWindowSize << "x" << YWindowSize;
    string windowSize = ss.str();

    // Setting up the movie buffer with the dynamic window size
    string baseCommand = "ffmpeg -loglevel quiet -r 60 -f rawvideo -pix_fmt rgba -s " + windowSize + " -i - "
                         "-c:v libx264rgb -threads 0 -preset fast -y -pix_fmt yuv420p -crf 0 -vf vflip 2>ffmpeg_error.log ";

    string z = baseCommand + ts;

    const char *ccx = z.c_str();
    MovieFile = popen(ccx, "w");

    // Check if popen was successful
    if (MovieFile == NULL) {
        fprintf(stderr, "Error: Failed to open movie file with popen\n");
        return;
    }

    // Allocate buffer
    Buffer = (int*)malloc(XWindowSize * YWindowSize * sizeof(int));

    // Check if malloc was successful
    if (Buffer == NULL) {
        fprintf(stderr, "Error: Failed to allocate memory for buffer\n");
        pclose(MovieFile);
        MovieFile = NULL;
        return;
    }

    MovieOn = 1;
    printf("Movie recording started successfully\n");
}

void movieOff()
{
    if (MovieOn == 1) 
    {
        pclose(MovieFile);
    }
    free(Buffer);
    MovieOn = 0;
    printf("Movie recording stopped successfully\n");
}

void screenShot()
{	
	int pauseFlag;
	FILE* ScreenShotFile;
	int* buffer;

	//convert the x and y windowsize to a string of format "XsizexYsize"
    stringstream ss;
    ss << XWindowSize << "x" << YWindowSize;
    string windowSize = ss.str();

    // Construct the ffmpeg command with the dynamic window size
    string baseCommand = "ffmpeg -loglevel quiet -framerate 60 -f rawvideo -pix_fmt rgba -s " + windowSize + " -i - "
                         "-c:v libx264rgb -threads 0 -preset fast -y -crf 0 -vf vflip output1.mp4";
    const char* cmd = baseCommand.c_str();
	//const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1000x1000 -i - "
	//              "-threads 0 -preset fast -y -pix_fmt yuv420p -crf 21 -vf vflip output1.mp4";
	ScreenShotFile = popen(cmd, "w");
	buffer = (int*)malloc(XWindowSize*YWindowSize*sizeof(int));
	
	if(Pause == 0) 
	{
		Pause = 1;
		pauseFlag = 0;
	}
	else
	{
		pauseFlag = 1;
	}
	
	for(int i =0; i < 1; i++)
	{
		drawPicture();
		glReadPixels(5, 5, XWindowSize, YWindowSize, GL_RGBA, GL_UNSIGNED_BYTE, buffer);
		fwrite(buffer, sizeof(int)*XWindowSize*YWindowSize, 1, ScreenShotFile);
	}
	
	pclose(ScreenShotFile);
	free(buffer);

	string ts = getTimeStamp(); // Only storing in a separate variable for debugging purposes.
	string s = "ffmpeg -loglevel quiet -i output1.mp4 -qscale:v 1 -qmin 1 -qmax 1 " + ts + ".jpeg";
	// Convert back to a C-style string.
	const char *ccx = s.c_str();
	system(ccx);
	system("rm output1.mp4");
	printf("\nScreenshot Captured: \n");
	cout << "Saved as " << ts << ".jpeg" << endl;
	
	Pause = pauseFlag;
	//ffmpeg -i output1.mp4 output_%03d.jpeg
}